#include "hip/hip_runtime.h"
﻿#include"functions.cuh"


void readdata(int N, hipfftComplex *data, string name, bool readbinary) {
	if (readbinary == true) {
		readdatabinary(N, data, name);
	}
	else {
		readdatatxt(N, data, name);
	}
}

void writedata(int N, hipfftComplex *data, string name, bool writebinary) {
	if (writebinary == true) {
		writedatabinary(N, data, name);
	}
	else {
		writedatatxt(N, data, name);
	}
}


void readdatabinary(int N, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	float num1,num2;
	
	if (myfile.is_open())
	{
		int k = 0;
		while (k < N)
		{
			
			myfile.read((char*)&num1, sizeof(num1));
			myfile.read((char*)&num2, sizeof(num2));
			data[k].x = num1;
			data[k].y =  num2;
			k++;
		}
		myfile.close();
	}
	else cout << "Unable to open file";
}


void readdatatxt(int N, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	string line;
	

	if (myfile.is_open())
	{
		int k = 0;
		while (k < N)
		{
			getline(myfile, line, '\n');
			data[k].x = stof(line);
			data[k].y = 0;

			k++;
		}
		myfile.close();
	}
	else cout << "Unable to open file";
}

void writedatatxt(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			 myfile<< ii << " " << (data1[ii].x)<< " " << data1[ii].y <<"\n"; 
	
		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writedatabinary(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			
			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writetime(int N, string name, long long *readtime, long long *shifttime, long long *ffttime,
	long long *multime, long long *ifftime, long long *writetime, long long *looptime) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		myfile << "Atempt\t\tReadT.\t\tShiftT.\t\tFFTT.\t\tMulT.\t\tIFFTT.\t\tWriteT.\t\tLoopT." << "\n";
		for (int ii = 0; ii < N; ii++)
		{
			myfile << ii << "\t\t" << readtime[ii] << "\t\t" << shifttime[ii] << "\t\t" << ffttime[ii] << "\t\t" << multime[ii];
			myfile << "\t\t" << ifftime[ii] << "\t\t" << writetime[ii] << "\t\t" << looptime[ii] << "\n";

		}
		myfile.close();
	}

	else cout << "Unable to open file";
}

void planfftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {


	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize - overlap, odist = fftsize;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void planifftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

/*Makes the complex conjugate of data2 and multiply point by point data1 and data2
**data1 and data2 should be on device memory!!
N: length of data
data1: hipfftComplex data set 1
data1: hipfftComplex data set 2*/
__global__ void multip(int n, hipfftComplex *data1, hipfftComplex *data2, int refsize)
{
	hipfftComplex aux;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		int k = i % refsize;
		
		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i

		aux.x = data1[i].x* data2[k].x - data1[i].y*(-data2[k].y);
		aux.y = data1[i].x*(-data2[k].y) + data1[i].y*data2[k].x;
	
		
		data1[i].x = aux.x;
		data1[i].y = aux.y;
	}
}

__global__ void extendRefSignal(int n, hipfftComplex *data, int refsize) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		
		if (i >= refsize) {
			data[i] = data[i%refsize];

		}
	}
}



__global__
void maskandshift(char *devicedata, hipComplex *Dcomplexdata, int N)
{
	unsigned k;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += stride) {

		k = (unsigned)(devicedata[i]);
		Dcomplexdata[i * 4 + 3].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 3].x = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 2].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 2].x = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 1].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 1].x = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 0].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 0].x = (k % 2);


	}
}

void readrealdata(int N, char *data, string name)
{

	ifstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.read(data, N);

		/*int k = 0;
		while (k < N)
		{
			k++
		}*/
		myfile.close();
	}
	else cout << "Unable to open file";
}

