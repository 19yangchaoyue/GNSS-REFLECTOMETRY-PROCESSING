#include "hip/hip_runtime.h"
﻿#include"functions.cuh"
#include "extra/TextParser.cuh"

void readConfig(const char *configFileName, int numofDataLines, int *fftsize, int *numofFFts, int *overlap, int *fSampling, int *blockSize, int *peakRangeStd, int *peakSamplesToSave,
	int* quantOfAverIncoh, int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames,string *fileRefNames) {

	TextParser t(configFileName);

	TextParserSafeCall(t.seek("*FFTSIZE"));
	*fftsize = t.getint();
	TextParserSafeCall(t.seek("*NUMOFFFTS"));
	*numofFFts = t.getint();
	TextParserSafeCall(t.seek("*QUANTINCOHAVER"));
	*quantOfAverIncoh = t.getint();
	TextParserSafeCall(t.seek("*OVERLAP"));
	*overlap = t.getint();
	TextParserSafeCall(t.seek("*FSAMPLING"));
	*fSampling = t.getint();
	TextParserSafeCall(t.seek("*BLOCKSIZE"));
	*blockSize = t.getint();
	TextParserSafeCall(t.seek("*PEAKRANGESTD"));
	*peakRangeStd = t.getint();
	TextParserSafeCall(t.seek("*PEAKSAMPLESTOSAVE"));
	*peakSamplesToSave = t.getint();

	TextParserSafeCall(t.seek("*QUANTDATALINES"));
	if (t.getint() != numofDataLines) {
		cout << "diferent num of Data lines in the file than declared on arguments \n  To execute enter arguments: NameconfigFile.ASE NumofDataLines\n";
		exit(1);
	}

	for (int i = 0; i < numofDataLines; i++) {
		TextParserSafeCall(t.seek("*DATALINE"));

		fileNames[i] = t.getword();
		dataOffsetBeg[i] = t.getint();
		dataOffsetEnd[i] = t.getint();
		doppler[i] = t.getint();
		fileRefNames[i] = t.getword();
	}
}

void checkInputConfig(int argc, const char **argv, int numofDataLines, int fftsize, int numofFFts, int overlap, int fSampling,  int blockSize, int peakRangeStd, int peakSamplesToSave,
	int quantOfAverIncoh,  int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames, string *fileRefNames) {

	if (argc != 3) {
		cout << "Error: Wrong number of arguments\n"; 
		exit(0);
	}

	cout << "\n" << "Quant of args: " << argc << "\n";
	cout << "First: " << argv[0] << "\n";
	cout << "Second: " << argv[1] << "\n";
	cout << "Third: " << argv[2] << "\n\n";

	cout << "FFT Size: " << fftsize << "\n";
	cout << "Num. of FFT: " << numofFFts << "\n";
	cout << "Overlap: " << overlap << "\n";
	cout << "FSampling: " << fSampling << "\n";
	cout << "Quant of averg Inch.: " << quantOfAverIncoh << "\n";
	cout << "Blok Size: " << blockSize << "\n";
	cout << "Peak samples for the std: " << peakRangeStd << "\n";
	cout << "Peak samples to save: " << peakSamplesToSave << "\n";

	cout << "Num of data lines: " << numofDataLines << "\n";
	cout << "Data lines: \n";
	for (int i = 0; i < numofDataLines; i++) {
		cout << fileNames[i] << "  ";
		cout << dataOffsetBeg[i] << "  ";
		cout << dataOffsetEnd[i] << "  ";
		cout << doppler[i] << " ";
		cout << fileRefNames[i] << "\n";

	}

}

void readdata(int length,int offsetFromBeg, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	float num1,num2;
	
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg*2 * sizeof(float));
		int k = 0;
		while (k < length)
		{
			
			myfile.read((char*)&num1, sizeof(num1));
			myfile.read((char*)&num2, sizeof(num2));
			data[k].x = num1;
			data[k].y =  num2;
			k++;
		}
		myfile.close();
	}
	else cout << "Unable to open file";
}

void readRealData(int length, int offsetFromBeg, int bytesToRead,char *data, string name)
{
	if (length > bytesToRead) {
		cout << "Error: iput length bigger than bytesToRead";
		exit(0);
	}

	ifstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg*sizeof(char));
		myfile.read(data, length);
				
		myfile.close();
		if(length< bytesToRead){
			cout << bytesToRead- length << "Warning: length smaller than bytesToRead, Bytes filled with 0 \n";
			memset(&data[length], 0, bytesToRead - length);
		}
	}
	else cout << "Unable to open file";
}


void writeIncoh(int N, hipComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N/2; ii++)
		{
			

			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
			

		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writeMaxs(int N, Npp32f *dataMaxValue, int *dataMaxPos, Npp32f *hostarrayStd,int doppler,string name) {

	ofstream myfile;
	myfile.open(name, ios::app);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			myfile <<"Pos: "<< dataMaxPos[ii]<<" Value: " << dataMaxValue[ii] << " STD: " << hostarrayStd[ii]<< " Doppler: " << doppler<< "\n";

		}
		myfile << "\n --------------------------------------------------------------------- \n\n";
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writedata(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			
			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writetime(int N, string name, long long *readtime, long long *writetime, long long *looptime) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		myfile << "Atempt\t\tReadT.\t\tWriteT.\t\tLoopT." << "\n";
		for (int ii = 0; ii < N; ii++)
		{
			myfile << ii << "\t\t" << readtime[ii] << "\t\t" << writetime[ii] << "\t\t" << looptime[ii] << "\n";

		}
		myfile.close();
	}

	else cout << "Unable to open file";
}

void planfftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {


	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize - overlap, odist = fftsize;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void planifftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void maxCompute(int numofIncoherentSums, Npp32f *deviceDataIncoherentSum, int fftsize, Npp32f *deviceArrayMaxs,
	 int *deviceArrayPos, Npp8u * pDeviceBuffer) {

	for (int i = 0; i < numofIncoherentSums; i++) {

		nppsMaxIndx_32f(&deviceDataIncoherentSum[i*fftsize], fftsize, &deviceArrayMaxs[i], &deviceArrayPos[i], pDeviceBuffer);
	}
}


void stdCompute(int numofIncoherentSums, Npp32f *dataIncoherentSum, int fftsize,
	Npp32f *deviceArraystd, int *arrayPos, Npp8u * pDeviceBuffer, int peakRange) {

	int leftPeakIndex, rightPeakIndex, stdLength;

	for (int i = 0; i < numofIncoherentSums; i++) {
		
		leftPeakIndex = arrayPos[i] - peakRange/2;
		rightPeakIndex = arrayPos[i] + peakRange/2;
		
		if (rightPeakIndex >= fftsize) {//case 2
			rightPeakIndex = rightPeakIndex % fftsize;
			stdLength = leftPeakIndex - rightPeakIndex;
			nppsStdDev_32f(&dataIncoherentSum[i*fftsize+ rightPeakIndex], stdLength, &deviceArraystd[i], pDeviceBuffer);
		}
		else if (leftPeakIndex < 0) {//case 3
			leftPeakIndex = fftsize + leftPeakIndex;
			stdLength = leftPeakIndex-rightPeakIndex ;
			nppsStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &deviceArraystd[i], pDeviceBuffer);
		}
		else {//case 1
			if (arrayPos[i] < fftsize / 2) {
				stdLength = fftsize- rightPeakIndex;
				nppsStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &deviceArraystd[i], pDeviceBuffer);
			}
			else {
				stdLength = leftPeakIndex;
				nppsStdDev_32f(&dataIncoherentSum[i*fftsize], stdLength, &deviceArraystd[i], pDeviceBuffer);
			}			
		}		
	}
}

__global__ void multip(int samples, hipfftComplex *data1, hipfftComplex *data2, int refsize)
{
	hipfftComplex aux;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		int k = i % refsize;
		
		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i

		aux.x = data1[i].x* data2[k].x - data1[i].y*(-data2[k].y);
		aux.y = data1[i].x*(-data2[k].y) + data1[i].y*data2[k].x;
	
		
		data1[i].x = aux.x;
		data1[i].y = aux.y;
	}
}

__global__ void extendRefSignal(int samples, hipfftComplex *data, int refsize) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		
		if (i >= refsize) {
			data[i] = data[i%refsize];

		}
	}
}


__global__ void applyDoppler(int samples, hipfftComplex *data, float freqDoppler, float fs, unsigned long long samplePhaseMantain)
{
	hipfftComplex aux, aux2;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		aux2.x=cos(2.0*PI*(i+ samplePhaseMantain)*(float(freqDoppler)/float(fs)));
		aux2.y= sin(2.0*PI*(i+ samplePhaseMantain)*(float(freqDoppler) / float(fs)));

		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i
		aux.x = data[i].x*aux2.x - data[i].y*aux2.y;
		aux.y = data[i].x*aux2.y + data[i].y*aux2.x;

		data[i].x= aux.x;
		data[i].y= aux.y;
		 
	
	}
}

__global__ void savePeak(int numOfFFT, hipfftComplex *dataFromIFFT, hipfftComplex *dataToSave, int peakSamplesToSave,
	int quantOfIncohSumAve,int fftsize, int *arrayPos) {

	int samplesToSave = numOfFFT * peakSamplesToSave;
	int posOnIFFT,fftOfThePeak,indexOfArrayPos, leftPosMaxOnOneIFFT, posOnOneIFFT;//rightPosMaxOnOneIFFT;
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesToSave; i += stride) {

		fftOfThePeak = i / peakSamplesToSave;//num of FFT in dataFromInv
		indexOfArrayPos = fftOfThePeak / quantOfIncohSumAve;//number of index in arrayPos
		//rightPosMaxOnOneIFFT = (arrayPos[indexOfPos] + peakSamplesToSave / 2);
		leftPosMaxOnOneIFFT = arrayPos[indexOfArrayPos] - (peakSamplesToSave / 2);//begining of data to save
		posOnOneIFFT = leftPosMaxOnOneIFFT + (i%peakSamplesToSave);// sample of i in one fft

		if (posOnOneIFFT >= fftsize) {
			posOnIFFT = fftOfThePeak * fftsize + posOnOneIFFT%fftsize; //sample in the data from IFFT
			dataToSave[i] = dataFromIFFT[posOnIFFT];
			
			//case 2
		}
		else if (posOnOneIFFT < 0) {
			posOnIFFT = fftOfThePeak * fftsize + (fftsize+posOnOneIFFT);//sample in the data from IFFT
			dataToSave[i] = dataFromIFFT[posOnIFFT];
			//case 3
		}
		else {
			posOnIFFT = fftOfThePeak * fftsize + posOnOneIFFT;
			dataToSave[i] = dataFromIFFT[posOnIFFT];//sample in the data from IFFT
			//case 1
		}
	}
}

__global__ void inchoerentSum(int samplesInchoerentSum, hipfftComplex *dataFromInv, Npp32f *dataStorageInocherentSum,
	int quantofAverageIncoherent, int fftsize)
{
	
	int indexofInv, numofSumM;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesInchoerentSum; i += stride) {
		dataStorageInocherentSum[i] = 0;
		numofSumM = i / fftsize;
		for (int k = 0; k < quantofAverageIncoherent; k++) {
			indexofInv = numofSumM*quantofAverageIncoherent*fftsize + k*fftsize+ i%fftsize;
			dataStorageInocherentSum[i] += dataFromInv[indexofInv].x*dataFromInv[indexofInv].x + dataFromInv[indexofInv].y*dataFromInv[indexofInv].y;
		}

	}
}


__global__ void scale(int samples, hipfftComplex *data, int fftsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		data[i].x = data[i].x / float(fftsize);
		data[i].y = data[i].y / float(fftsize);

	}
}

__global__ void maskAndShift(char *devicedata, hipComplex *Dcomplexdata, int totalBytes)
{
	unsigned k;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < totalBytes; i += stride) {

		k = (unsigned)(devicedata[i]);
		Dcomplexdata[i * 4 + 3].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 3].x = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 2].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 2].x = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 1].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 1].x = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 0].y = (k % 2);
		k = k / 2;
		Dcomplexdata[i * 4 + 0].x = (k % 2);


	}
}
