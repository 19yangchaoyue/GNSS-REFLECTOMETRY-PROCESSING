#include "hip/hip_runtime.h"
﻿#include"functions.cuh"
#include "extra/TextParser.cuh"

void readConfig(const char *configFileName, int numofDataLines, int *fftsize, int *numofFFts, int *overlap, int *fSampling, int* quantOfAverIncoh
	, bool *readbinary, bool *writebinary, int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames) {

	TextParser t(configFileName);

	TextParserSafeCall(t.seek("*FFTSIZE"));
	*fftsize = t.getint();
	TextParserSafeCall(t.seek("*NUMOFFFTS"));
	*numofFFts = t.getint();
	TextParserSafeCall(t.seek("*QUANTINCOHAVER"));
	*quantOfAverIncoh = t.getint();
	TextParserSafeCall(t.seek("*OVERLAP"));
	*overlap = t.getint();
	TextParserSafeCall(t.seek("*FSAMPLING"));
	*fSampling = t.getint();
	TextParserSafeCall(t.seek("*BOOLREADBIN"));
	*readbinary = t.getint();
	TextParserSafeCall(t.seek("*BOOLWRITEBIN"));
	*writebinary = t.getint();

	TextParserSafeCall(t.seek("*QUANTDATALINES"));
	if (t.getint() != numofDataLines) {
		cout << "diferent num of Data lines in the file than declared on arguments \n  To execute enter arguments: NameconfigFile.ASE NumofDataLines\n";
		exit(1);
	}

	for (int i = 0; i < numofDataLines; i++) {
		TextParserSafeCall(t.seek("*DATALINE"));

		fileNames[i] = t.getword();
		dataOffsetBeg[i] = t.getint();
		dataOffsetEnd[i] = t.getint();
		doppler[i] = t.getint();
	}
}

void checkInputConfig(int argc, const char **argv, int numofDataLines, int fftsize, int numofFFts, int overlap, int fSampling, int quantOfAverIncoh
	, bool readbinary, bool writebinary, int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames) {

	cout << "\n" << "Quant of args: " << argc << "\n";
	cout << "First: " << argv[0] << "\n";
	cout << "Second: " << argv[1] << "\n";
	cout << "Third: " << argv[2] << "\n\n";

	cout << numofDataLines << "\n";
	cout << fftsize << "\n";
	cout << numofFFts << "\n";
	cout << overlap << "\n";
	cout << fSampling << "\n";
	cout << quantOfAverIncoh << "\n";
	cout << readbinary << "\n";
	cout << writebinary << "\n";

	for (int i = 0; i < numofDataLines; i++) {
		cout << fileNames[i] << "  ";
		cout << dataOffsetBeg[i] << "  ";
		cout << dataOffsetEnd[i] << "  ";
		cout << doppler[i] << "\n";

	}

}


void readdata(int N, hipfftComplex *data, string name, bool readbinary) {
	if (readbinary == true) {
		readdatabinary(N,0, data, name);
	}
	else {
		readdatatxt(N, data, name);
	}
}

void writedata(int length, hipfftComplex *data, string name, bool writebinary) {
	if (writebinary == true) {
		writedatabinary(length, data, name);
	}
	else {
		writedatatxt(length, data, name);
	}
}


void readdatabinary(int length,int offsetFromBeg, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	float num1,num2;
	
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg*2 * sizeof(float));
		int k = 0;
		while (k < length)
		{
			
			myfile.read((char*)&num1, sizeof(num1));
			myfile.read((char*)&num2, sizeof(num2));
			data[k].x = num1;
			data[k].y =  num2;
			k++;
		}
		myfile.close();
	}
	else cout << "Unable to open file";
}


void readdatatxt(int N, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	string line;
	

	if (myfile.is_open())
	{
		int k = 0;
		while (k < N)
		{
			getline(myfile, line, '\n');
			data[k].x = stof(line);
			data[k].y = 0;

			k++;
		}
		myfile.close();
	}
	else cout << "Unable to open file";
}

void writedatatxt(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			myfile<< ii << " " << (data1[ii].x)<< " " << data1[ii].y <<"\n"; 
			//myfile << data1[ii].x << "\n";
			//myfile << data1[ii].y << "\n";
	
		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writeIncohtxt(int N, hipComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N/2; ii++)
		{
			
			myfile << data1[ii].x << "\n";
			myfile << data1[ii].y << "\n";
			

		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writeMaxstxt(int N, Npp32f *dataMaxValue, int *dataMaxPos, Npp32f *hostarrayStd,string name) {

	ofstream myfile;
	myfile.open(name, ios::app);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			myfile <<"Pos: "<< dataMaxPos[ii]<<" Value: " << dataMaxValue[ii] << " STD: " << hostarrayStd[ii] << "\n";

		}
		myfile << "\n --------------------------------------------------------------------- \n\n";
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writedatabinary(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			
			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}
		myfile.close();
	}

	else cout << "Unable to open file\n";
}

void writetime(int N, string name, long long *readtime, long long *shifttime, long long *ffttime,
	long long *multime, long long *ifftime, long long *writetime, long long *looptime) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		myfile << "Atempt\t\tReadT.\t\tShiftT.\t\tFFTT.\t\tMulT.\t\tIFFTT.\t\tWriteT.\t\tLoopT." << "\n";
		for (int ii = 0; ii < N; ii++)
		{
			myfile << ii << "\t\t" << readtime[ii] << "\t\t" << shifttime[ii] << "\t\t" << ffttime[ii] << "\t\t" << multime[ii];
			myfile << "\t\t" << ifftime[ii] << "\t\t" << writetime[ii] << "\t\t" << looptime[ii] << "\n";

		}
		myfile.close();
	}

	else cout << "Unable to open file";
}

void planfftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {


	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize - overlap, odist = fftsize;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void planifftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void maxAndStd(int numofIncoherentSums, Npp32f *dataIncoherentSum, int fftsize, Npp32f *arrayMaxs,
	Npp32f *arraystd, int *arrayPos, Npp8u * pDeviceBuffer) {



	for (int i = 0; i < numofIncoherentSums; i++) {


		nppsMaxIndx_32f(&dataIncoherentSum[i*fftsize], fftsize, &arrayMaxs[i], &arrayPos[i], pDeviceBuffer);
		nppsStdDev_32f(&dataIncoherentSum[i*fftsize], fftsize, &arraystd[i], pDeviceBuffer);

	}
	hipDeviceSynchronize();




}

/*Makes the complex conjugate of data2 and multiply point by point data1 and data2
**data1 and data2 should be on device memory!!
N: length of data
data1: hipfftComplex data set 1
data1: hipfftComplex data set 2*/
__global__ void multip(int samples, hipfftComplex *data1, hipfftComplex *data2, int refsize)
{
	hipfftComplex aux;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		int k = i % refsize;
		
		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i

		aux.x = data1[i].x* data2[k].x - data1[i].y*(-data2[k].y);
		aux.y = data1[i].x*(-data2[k].y) + data1[i].y*data2[k].x;
	
		
		data1[i].x = aux.x;
		data1[i].y = aux.y;
	}
}

__global__ void extendRefSignal(int samples, hipfftComplex *data, int refsize) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		
		if (i >= refsize) {
			data[i] = data[i%refsize];

		}
	}
}


__global__ void applyDoppler(int samples, hipfftComplex *data, float freq, float fs,int samplePhaseMantain)
{
	hipfftComplex aux, aux2;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		aux2.x=cos(2.0*PI*(i+ samplePhaseMantain)*(float(freq)/float(fs)));
		aux2.y= sin(2.0*PI*(i+ samplePhaseMantain)*(float(freq) / float(fs)));

		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i
		aux.x = data[i].x*aux2.x - data[i].y*aux2.y;
		aux.y = data[i].x*aux2.y + data[i].y*aux2.x;

		data[i].x= aux.x;
		data[i].y= aux.y;
		 
	
	}
}


__global__ void inchoerentSum(int samplesInchoerentSum, hipfftComplex *dataFromInv, Npp32f *dataStorageInocherentSum,
	int quantofAverageIncoherent, int fftsize)
{
	
	int indexofInv, numofSumM;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesInchoerentSum; i += stride) {
		dataStorageInocherentSum[i] = 0;
		numofSumM = i / fftsize;
		for (int k = 0; k < quantofAverageIncoherent; k++) {
			indexofInv = numofSumM*quantofAverageIncoherent*fftsize + k*fftsize+ i%fftsize;
			dataStorageInocherentSum[i] += dataFromInv[indexofInv].x*dataFromInv[indexofInv].x + dataFromInv[indexofInv].y*dataFromInv[indexofInv].y;
		}

	}
}


__global__ void scale(int samples, hipfftComplex *data, int fftsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		data[i].x = data[i].x / float(fftsize);
		data[i].y = data[i].y / float(fftsize);

	}
}


