#include "hip/hip_runtime.h"
﻿#include"functions.cuh"
#include "extra/TextParser.cuh"

void readConfig(const char *configFileName, int numofDataLines, int *fftsize, int *numofFFts, int *overlap, int *fSampling, int *blockSize, int *peakRangeStd, int *peakSamplesToSave,
	int* quantOfAverIncoh, int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames,string *fileRefNames, int *ddmspan, int *ddmnumdiv) {

	TextParser t(configFileName);

	TextParserSafeCall(t.seek("*FFTSIZE"));
	*fftsize = t.getint();
	TextParserSafeCall(t.seek("*NUMOFFFTS"));
	*numofFFts = t.getint();
	TextParserSafeCall(t.seek("*QUANTINCOHAVER"));
	*quantOfAverIncoh = t.getint();
	TextParserSafeCall(t.seek("*OVERLAP"));
	*overlap = t.getint();
	TextParserSafeCall(t.seek("*FSAMPLING"));
	*fSampling = t.getint();
	TextParserSafeCall(t.seek("*BLOCKSIZE"));
	*blockSize = t.getint();
	TextParserSafeCall(t.seek("*PEAKRANGESTD"));
	*peakRangeStd = t.getint();
	TextParserSafeCall(t.seek("*PEAKSAMPLESTOSAVE"));
	*peakSamplesToSave = t.getint();
	TextParserSafeCall(t.seek("*REFFILENAME"));
	*fileRefNames = t.getword();
	TextParserSafeCall(t.seek("*DDMFREQSPAN"));
	*ddmspan = t.getint();
	TextParserSafeCall(t.seek("*DDMNUMDIVISIONS"));
	*ddmnumdiv = t.getint();


	TextParserSafeCall(t.seek("*QUANTDATALINES"));
	if (t.getint() != numofDataLines) {
		cout << "diferent num of Data lines in the file than declared on arguments \n  To execute enter arguments: NameconfigFile.ASE NumofDataLines\n";
		exit(1);
	}

	for (int i = 0; i < numofDataLines; i++) {
		TextParserSafeCall(t.seek("*DATALINE"));

		fileNames[i] = t.getword();
		dataOffsetBeg[i] = t.getint();
		dataOffsetEnd[i] = t.getint();
		doppler[i] = t.getint();
		
	}
}

void checkInputConfig(int argc, const char **argv, int numofDataLines, int fftsize, int numofFFts, int overlap, int fSampling,  int blockSize, int peakRangeStd, int peakSamplesToSave,
	int quantOfAverIncoh,  int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames, string fileRefNames, int ddmspan, int ddmnumdiv) {

	if (argc != 3) {
		cout << "Error: Wrong number of arguments\n"; 
		exit(0);
	}

	cout << "\n" << "Quant of args: " << argc << "\n";
	cout << "First: " << argv[0] << "\n";
	cout << "Second: " << argv[1] << "\n";
	cout << "Third: " << argv[2] << "\n\n";

	cout << "FFT Size: " << fftsize << "\n";
	cout << "Num. of FFT: " << numofFFts << "\n";
	cout << "Overlap: " << overlap << "\n";
	cout << "FSampling: " << fSampling << "\n";
	cout << "Quant of averg Inch.: " << quantOfAverIncoh << "\n";
	cout << "Blok Size: " << blockSize << "\n";
	cout << "Peak samples for the std: " << peakRangeStd << "\n";
	cout << "Peak samples to save: " << peakSamplesToSave << "\n";
	cout << "Ref File Name: " << fileRefNames << "\n";
	cout << "DDM span: " << ddmspan << "\n";
	cout << "DDM num of div: " << ddmnumdiv << "\n";


	cout << "Num of data lines: " << numofDataLines << "\n";
	cout << "Data lines: \n";
	for (int i = 0; i < numofDataLines; i++) {
		cout << fileNames[i] << "  ";
		cout << dataOffsetBeg[i] << "  ";
		cout << dataOffsetEnd[i] << "  ";
		cout << doppler[i] << "\n";

	}

}

void readdata(int length,int offsetFromBeg, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	//float num1,num2;
	
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg* sizeof(hipfftComplex));
		
		myfile.read((char*)data, length*sizeof(hipfftComplex));
		/*int k = 0;
		while (k < length)
		{
			
			myfile.read((char*)&num1, sizeof(num1));
			myfile.read((char*)&num2, sizeof(num2));
			data[k].x = num1;
			data[k].y =  num2;
			k++;
		}*/
		myfile.close();
	}
	else cout << "Unable to open file of floats for reading " << name << "\n";
}

void readRealData(int length, int offsetFromBeg, int bytesToRead,char *data, string name)
{
	if (length > bytesToRead) {
		cout << "Error: iput length bigger than bytesToRead\n";
		exit(0);
	}

	ifstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg*sizeof(char));
		myfile.read(data, length);
				
		myfile.close();
		if(length< bytesToRead){
			cout << "Warning: length smaller than bytesToRead, " << bytesToRead - length<<" Bytes filled with 0 \n Last/s incoherents will be incomplete \n";
			memset(&data[length], 0, bytesToRead - length);
		}
	}
	else cout << "Unable to open file of Real Data for reading " << name << "\n";
}


void writeIncoh(int N, hipComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N/2; ii++)
		{
			

			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
			

		}
		myfile.close();
	}

	else cout << "Unable to open file of incoh for writting " << name << "\n";
}

void writeMaxs(int N, Npp32f *dataMaxValue, int *dataMaxPos, Npp32f *hostarrayStd,int doppler,string name) {

	ofstream myfile;
	myfile.open(name, ios::app);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			myfile <<"Pos: "<< dataMaxPos[ii]<<" Value: " << dataMaxValue[ii] << " STD: " << hostarrayStd[ii]<< " Doppler: " << doppler<< "\n";

		}
		myfile << "\n --------------------------------------------------------------------- \n\n";
		myfile.close();
	}

	else cout << "Unable to open file of Maxs " << name << "\n";
}

void writedata(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{


		myfile.write((char*)data1, N*sizeof(hipfftComplex));
		
		/*for (int ii = 0; ii < N; ii++)
		{
			
			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}*/
		myfile.close();
	}

	else cout << "Unable to open file of data to write " << name << "\n";
}

void writetime(int N, string name, long long *readtime, long long *writetime, long long *looptime
	, long long *mask_elapsed_secs, long long *doppler_elapsed_secs, 
	long long *fft_elapsed_secs, long long *mult_elapsed_secs, long long *ifft_elapsed_secs, long long *scale_elapsed_secs,
	long long *incho_elapsed_secs, long long *max_elapsed_secs, long long *savep_elapsed_secs, long long *std_elapsed_secs) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		myfile << "Atempt\t\tReadT.\t\tMask\t\tDoppler\t\tFFT\t\tMul\t\tIFFT\t\tScale\t\tIncoh\t\tMax\t\tSaveP.\t\tSTD\t\tWriteT.\t\tLoopT." << "\n";
		for (int ii = 0; ii < N; ii++)
		{
			myfile << ii << "\t\t" << readtime[ii] << "\t\t" << mask_elapsed_secs[ii] << "\t\t"
				<< doppler_elapsed_secs[ii] << "\t\t"
				 << fft_elapsed_secs[ii] << "\t\t" << mult_elapsed_secs[ii] << "\t\t"
				 << ifft_elapsed_secs[ii] << "\t\t" << scale_elapsed_secs[ii] << "\t\t"
				 << incho_elapsed_secs[ii] << "\t\t" << max_elapsed_secs[ii] << "\t\t"
				 << savep_elapsed_secs[ii] << "\t\t" << std_elapsed_secs[ii] << "\t\t"
				<< writetime[ii] << "\t\t" << looptime[ii] << "\n";

		}
		myfile.close();
	}

	else cout << "Unable to open file of times "<<name<<"\n";
}

void planfftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {


	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize - overlap, odist = fftsize;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void planifftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void maxCompute(int numofIncoherentSums, Npp32f *deviceDataIncoherentSum, int fftsize, Npp32f *deviceArrayMaxs,
	 int *deviceArrayPos, Npp8u * pDeviceBuffer) {

	for (int i = 0; i < numofIncoherentSums; i++) {

		nppsMaxIndx_32f(&deviceDataIncoherentSum[i*fftsize], fftsize, &deviceArrayMaxs[i], &deviceArrayPos[i], pDeviceBuffer);
	}
}


void stdCompute(int numofIncoherentSums, Npp32f *dataIncoherentSum, int fftsize,
	Npp32f *deviceArraystd, int *arrayPos, Npp8u * pDeviceBuffer, int peakRange) {

	int leftPeakIndex, rightPeakIndex, stdLength;
	stdLength = (fftsize / 2) - ((peakRange) / 2)-1;
	for (int i = 0; i < numofIncoherentSums; i++) {
		
		leftPeakIndex = arrayPos[i] - peakRange/2;
		rightPeakIndex = arrayPos[i] + peakRange/2;
		
		if (rightPeakIndex >= fftsize) {//case 2
			rightPeakIndex = rightPeakIndex % fftsize;
			//stdLength = leftPeakIndex - rightPeakIndex;
			nppsStdDev_32f(&dataIncoherentSum[i*fftsize+ rightPeakIndex], stdLength, &deviceArraystd[i], pDeviceBuffer);
		}
		else if (leftPeakIndex < 0) {//case 3
			leftPeakIndex = fftsize + leftPeakIndex;
			//stdLength = leftPeakIndex-rightPeakIndex ;
			nppsStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &deviceArraystd[i], pDeviceBuffer);
		}
		else {//case 1
			if (arrayPos[i] < fftsize / 2) {
				//stdLength = fftsize- rightPeakIndex;
				nppsStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &deviceArraystd[i], pDeviceBuffer);
			}
			else {
				//stdLength = leftPeakIndex;
				nppsStdDev_32f(&dataIncoherentSum[i*fftsize], stdLength, &deviceArraystd[i], pDeviceBuffer);
			}			
		}		
	}
}

__global__ void multip(int samples, hipfftComplex *data1, hipfftComplex *data2, int refsize)
{
	hipfftComplex aux;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		int k = i % refsize;
		
		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i

		aux.x = data1[i].x* data2[k].x - data1[i].y*(-data2[k].y);
		aux.y = data1[i].x*(-data2[k].y) + data1[i].y*data2[k].x;
	
		
		data1[i].x = aux.x;
		data1[i].y = aux.y;
	}
}

__global__ void extendRefSignal(int samples, hipfftComplex *data, int refsize) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		
		if (i >= refsize) {
			data[i] = data[i%refsize];

		}
	}
}


__global__ void applyDoppler(int samples, hipfftComplex *data, float freqDoppler, float fs, unsigned long long samplePhaseMantain)
{
	hipfftComplex aux, aux2;
	float angle;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		angle = 2.0*PI*float(i + samplePhaseMantain)*((freqDoppler) / (fs));
		aux2.x=cos(angle);
		aux2.y= sin(angle);
		
		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i
		aux.x = data[i].x*aux2.x - data[i].y*aux2.y;
		aux.y = data[i].x*aux2.y + data[i].y*aux2.x;

		data[i].x= aux.x;
		data[i].y= aux.y;
		 
	
	}
}

__global__ void savePeak(int numOfFFT, hipfftComplex *dataFromIFFT, hipfftComplex *dataToSave, int peakSamplesToSave,
	int quantOfIncohSumAve,int fftsize, int *arrayPos) {

	int samplesToSave = numOfFFT * peakSamplesToSave;
	int posOnIFFT,fftOfThePeak,indexOfArrayPos, leftPosMaxOnOneIFFT, posOnOneIFFT;//rightPosMaxOnOneIFFT;
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesToSave; i += stride) {

		fftOfThePeak = i / peakSamplesToSave;//num of FFT in dataFromInv
		indexOfArrayPos = fftOfThePeak / quantOfIncohSumAve;//number of index in arrayPos
		//rightPosMaxOnOneIFFT = (arrayPos[indexOfPos] + peakSamplesToSave / 2);
		leftPosMaxOnOneIFFT = arrayPos[indexOfArrayPos] - (peakSamplesToSave / 2);//begining of data to save
		posOnOneIFFT = leftPosMaxOnOneIFFT + (i%peakSamplesToSave);// sample of i in one fft

		if (posOnOneIFFT >= fftsize) {
			posOnIFFT = fftOfThePeak * fftsize + posOnOneIFFT%fftsize; //sample in the data from IFFT
			dataToSave[i] = dataFromIFFT[posOnIFFT];
			
			//case 2
		}
		else if (posOnOneIFFT < 0) {
			posOnIFFT = fftOfThePeak * fftsize + (fftsize+posOnOneIFFT);//sample in the data from IFFT
			dataToSave[i] = dataFromIFFT[posOnIFFT];
			//case 3
		}
		else {
			posOnIFFT = fftOfThePeak * fftsize + posOnOneIFFT;
			dataToSave[i] = dataFromIFFT[posOnIFFT];//sample in the data from IFFT
			//case 1
		}
	}
}

__global__ void inchoerentSum(int samplesInchoerentSum, hipfftComplex *dataFromInv, Npp32f *dataStorageInocherentSum,
	int quantofAverageIncoherent, int fftsize)
{
	
	int indexofInv, numofSumM;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesInchoerentSum; i += stride) {
		dataStorageInocherentSum[i] = 0;
		numofSumM = i / fftsize;
		for (int k = 0; k < quantofAverageIncoherent; k++) {
			indexofInv = numofSumM*quantofAverageIncoherent*fftsize + k*fftsize+ i%fftsize;
			dataStorageInocherentSum[i] += dataFromInv[indexofInv].x*dataFromInv[indexofInv].x + dataFromInv[indexofInv].y*dataFromInv[indexofInv].y;
		}

	}
}


__global__ void scale(int samples, hipfftComplex *data, int fftsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		data[i].x = data[i].x / float(fftsize);
		data[i].y = data[i].y / float(fftsize);

	}
}

__global__ void maskAndShift(char *devicedata, hipComplex *Dcomplexdata, int totalBytes)
{
	unsigned char k, aux;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < totalBytes; i += stride) {
		/*
		k = (unsigned char)(devicedata[i]);
		Dcomplexdata[i * 4 + 0].x = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 0].y = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 1].x = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 1].y = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 2].x = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 2].y = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 3].x = 2 * (k % 2) - 1;
		k = k / 2;
		Dcomplexdata[i * 4 + 3].y = 2 * (k % 2) - 1;
		*/
		k = (unsigned char)(devicedata[i]);
		
		aux = k & ((unsigned) 1);
		aux = aux >> 0;
		Dcomplexdata[i * 4 + 0].x = float(2 * (aux) - 1);


		aux = k & ((unsigned)(1<<1));
		aux = aux >> 1;

		
		Dcomplexdata[i * 4 + 0].y = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 2));
		aux = aux >> 2;
		Dcomplexdata[i * 4 + 1].x = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 3));
		aux = aux >> 3;
		Dcomplexdata[i * 4 + 1].y = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 4));
		aux = aux >> 4;
		Dcomplexdata[i * 4 + 2].x = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 5));
		aux = aux >> 5;
		Dcomplexdata[i * 4 + 2].y = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 6));
		aux = aux >> 6;
		Dcomplexdata[i * 4 + 3].x = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 7));
		aux = aux >> 7;
		Dcomplexdata[i * 4 + 3].y = float(2 * (aux)-1);





	}
}
