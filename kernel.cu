﻿#ifndef LIBRARIES
#define LIBRARIES
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <chrono>
#include <npps.h>
#define PI 3.14159265
using namespace std;
#endif

#include"functions.cuh"

int main(int argc, const char* argv[]) {
	hipDeviceReset();//reset device
	
	//READ CONFIG FILE
	int fftsize, fSampling, numofFFTs, overlap, quantofAverageIncoherent;
	bool readbinary = 1, writebinary = 0;
	int const numofDataLines = atoi(argv[2]);//substitut d'iterations
	string *fileNames;
	int *dataOffsetBeg, *dataOffsetEnd;
	int *doppler;

	fileNames = new string[numofDataLines];
	dataOffsetBeg = new int[numofDataLines];
	dataOffsetEnd = new int[numofDataLines];
	doppler = new int[numofDataLines];

	readConfig(argv[1], numofDataLines, &fftsize, &numofFFTs, &overlap, &fSampling, &quantofAverageIncoherent, &readbinary, &writebinary, dataOffsetBeg, dataOffsetEnd, doppler, fileNames);
	checkInputConfig(argc, argv, numofDataLines, fftsize, numofFFTs, overlap, fSampling, quantofAverageIncoherent, readbinary, writebinary, dataOffsetBeg, dataOffsetEnd, doppler, fileNames);

	//OTHER DECLARATIONS
	int samplesOfSignal = (numofFFTs * (fftsize-overlap))+overlap;//samples of complex data
	int bytesToRead = samplesOfSignal/4;
	if (samplesOfSignal % 4 != 0) { cout << "Warning bytesToRead rounded toward negative infinity: samplesOfSignal%4!=0 \n "; }
	int samplesWithOverlap= numofFFTs * fftsize;//total samples needed
	if(samplesOfSignal > samplesWithOverlap){ samplesWithOverlap = samplesOfSignal;}
	int inchoerentNumofFFT = numofFFTs/ quantofAverageIncoherent;

	int blockSize = 1024;//threads per block
	int numBlocks, nBufferSize, samplePhaseMantain,i;
	
	char *hostBytesOfData, *deviceBytesOfData;
	int *devicearrayPos,*hostarrayPos;
	hipfftComplex *deviceDataFile1, *deviceDataFile2, *hostDataFile1, *hostDataFile2;
	Npp32f *deviceIncoherentSum, *devicearrayMaxs, *devicearrayStd, *hostarrayMaxs, *hostarrayStd;
	Npp8u * pDeviceBuffer;
	
	int const maxIter = 5;
	long long read_elapsed_secs[maxIter], fft_elapsed_secs[maxIter], mul_elapsed_secs[maxIter], ifft_elapsed_secs[maxIter],write_elapsed_secs[maxIter], elapsed_secs[maxIter], shift_elapsed_secs[maxIter];
	
	//ALLOCATE
	hostBytesOfData = (char *)malloc(sizeof(char) * bytesToRead);
	hostarrayPos = new int[inchoerentNumofFFT];
	hostarrayMaxs = new Npp32f[inchoerentNumofFFT];
	hostarrayStd = new Npp32f[inchoerentNumofFFT];
	hostDataFile1 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * samplesWithOverlap);
	hostDataFile2 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * fftsize);
	CudaSafeCall(hipMalloc(&deviceBytesOfData, sizeof(char)*bytesToRead));
	CudaSafeCall(hipMalloc(&deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap));
	CudaSafeCall(hipMalloc(&deviceDataFile2, sizeof(hipfftComplex)*fftsize));
	CudaSafeCall(hipMalloc(&deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize));
	CudaSafeCall(hipMalloc(&devicearrayPos, sizeof(int)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT));
	nppsSumGetBufferSize_32f(fftsize, &nBufferSize);
	CudaSafeCall(hipMalloc((void **)(&pDeviceBuffer), nBufferSize));
	hipDeviceSynchronize();
	
	//FFT&IFFT PLANS
	hipfftHandle plan;
	hipfftHandle planref;
	hipfftHandle inverseplan;
	planfftFunction(fftsize, numofFFTs, overlap, &plan);
	planfftFunction(fftsize, 1, 0, &planref);
	planifftFunction(fftsize, numofFFTs, 0, &inverseplan);

	//LOOP
	for (i = 0; i < numofDataLines; i++) {
		auto Begin = std::chrono::high_resolution_clock::now();

		//READ DATA
		auto readdataBeg = chrono::high_resolution_clock::now();
		//readdata(samplesOfSignal, hostDataFile1, fileNames[i], readbinary);
		readdatabinary(dataOffsetEnd[i]-dataOffsetBeg[i], dataOffsetBeg[i], hostDataFile1, fileNames[i]);
		readdata(fftsize - overlap, hostDataFile2, "prn_L1CA_32_100.bin", readbinary);


		/*if (dataOffsetEnd[i] - dataOffsetBeg[i] > bytesToRead) { cout << "Indices of reading in config file exceed bytesToRead decleared"; }
		if ((dataOffsetEnd[i] - dataOffsetBeg[i])%(fftsize*quantofAverageIncoherent)!=0) {
		cout << "Warning length of data won't complete last incho sum in DATALINE: "<<i<<"\n"; }
		readRealData(dataOffsetEnd[i] - dataOffsetBeg[i], dataOffsetBeg[i],bytesToRead, hostBytesOfData, fileNames[i]);*/
		auto elapsed_read = chrono::high_resolution_clock::now() - readdataBeg;

		//CHECK: READED DATA 
		//writedata(samplesOfSignal/2, hostDataFile1, "rawsin.txt", writebinary);
		//writedata(fftsize- overlap, hostDataFile2, "rawsin2.txt", writebinary);

		//MEMORY FROM HOST TO DEVICE
		CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
		CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
		
		/*CudaSafeCall(hipMemcpy(deviceBytesOfData, hostBytesOfData, sizeof(char)*bytesToRead, hipMemcpyHostToDevice));*/
		hipDeviceSynchronize();
		
		//MASK AND SHIFT
		/*numBlocks = (bytesToRead + blockSize - 1) / blockSize;
		maskAndShift << <numBlocks, blockSize >> > (deviceBytesOfData, deviceDataFile1, bytesToRead);
		CudaCheckError();
		hipDeviceSynchronize();
		*/
		//MULTIPLY BY DOPPLER
		samplePhaseMantain = (i * fftsize*numofFFTs)%fSampling;
		numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		applyDoppler << <numBlocks, blockSize >> > (samplesOfSignal, deviceDataFile1, doppler[i], fSampling, samplePhaseMantain);
		CudaCheckError();
		hipDeviceSynchronize();
	
		//CHECK: doppler (only for printing doppler)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesOfSignal/2, hostDataFile1, "dopplerout.txt", writebinary);
		
		//EXTEND REFERENCE SIGNAL
		numBlocks = (fftsize + blockSize - 1) / blockSize;
		auto shiftBeg = chrono::high_resolution_clock::now();
		extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
		CudaCheckError();
		hipDeviceSynchronize();
		auto elapsed_shift = chrono::high_resolution_clock::now() - shiftBeg;

		//FFT
		auto fftBeg = chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(plan, deviceDataFile1, deviceDataFile1, HIPFFT_FORWARD));
		cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));
		hipDeviceSynchronize();
		auto elapsed_fft = chrono::high_resolution_clock::now() - fftBeg;

		//CHECK: FFT (only for printing fft)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "fft.txt", writebinary);

		//COMPLEX CONJUGATE AND MULTIPLICATION
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;
		auto mulBeg = chrono::high_resolution_clock::now();
		multip << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, deviceDataFile2, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		auto elapsed_mul = chrono::high_resolution_clock::now() - mulBeg;

		//CHECK: MULTIPLICATION (only for printing multiplication result)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "mult.txt", writebinary);
		
		//IFFT (To obtain original again it has to be devided for the # of elements)
		auto ifftBeg = chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(inverseplan, deviceDataFile1, deviceDataFile1, HIPFFT_BACKWARD));
		hipDeviceSynchronize();
		auto elapsed_ifft = chrono::high_resolution_clock::now() - ifftBeg;
		
		//SCALE
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;		
		scale << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();

		//INCOHERENT SUM
		numBlocks = (inchoerentNumofFFT*fftsize + blockSize - 1) / blockSize;
		inchoerentSum << <numBlocks, blockSize >> > (inchoerentNumofFFT*fftsize, deviceDataFile1, deviceIncoherentSum, quantofAverageIncoherent, fftsize);
		CudaCheckError(); 
		hipDeviceSynchronize();

		//MAXIMUM AND STD
		maxAndStd(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayMaxs, devicearrayStd,devicearrayPos, pDeviceBuffer);

		//CHECK: IFFT OR incho (not both at the same time)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize, hipMemcpyDeviceToHost)); //TO PRINT INCHO SUM
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost)); //TO PRINT IFFT RESULT
		//hipDeviceSynchronize();
		//writeIncohtxt(inchoerentNumofFFT*fftsize, hostDataFile1, "incoh.txt");//TO PRINT INCHO SUM
		//writedata(samplesWithOverlap, hostDataFile1,  "result.txt", writebinary);//TO PRINT IFFT RESULT 


		//MEMORY FROM HOST TO DEVICE FOR OUTPUT
		CudaSafeCall(hipMemcpy(hostarrayMaxs, devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayStd, devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayPos, devicearrayPos, sizeof(int)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		
		//OUTPUT
		auto writeBeg = chrono::high_resolution_clock::now();
		writeMaxstxt(inchoerentNumofFFT, hostarrayMaxs, hostarrayPos, hostarrayStd, "Maximums.txt");

		//ELAPSED TIME
		auto elapsed_write = chrono::high_resolution_clock::now() - writeBeg;
		auto elapsed_total = chrono::high_resolution_clock::now() - Begin;

		read_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_read).count();
		shift_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_shift).count();
		fft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_fft).count();
		mul_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_mul).count();
		ifft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_ifft).count();
		write_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_write).count();
		elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_total).count();
	}

	writetime(numofDataLines, "Times_op3.txt", read_elapsed_secs, shift_elapsed_secs, fft_elapsed_secs,
		mul_elapsed_secs, ifft_elapsed_secs, write_elapsed_secs, elapsed_secs);

	//FREE MEMORY
	cufftSafeCall(hipfftDestroy(plan));
	cufftSafeCall(hipfftDestroy(planref));
	cufftSafeCall(hipfftDestroy(inverseplan));
	hipFree(deviceDataFile1);
	hipFree(deviceDataFile2);
	hipFree(deviceIncoherentSum);
	hipFree(devicearrayPos);
	hipFree(deviceBytesOfData);
	hipFree(devicearrayMaxs);
	hipFree(pDeviceBuffer);
	hipFree(devicearrayStd);
	hipDeviceReset();
	delete[] fileNames;
	delete[] hostBytesOfData;
	delete[] hostarrayPos;
	delete[] hostarrayMaxs;
	delete[] hostarrayStd;
	delete[] hostDataFile2;
	delete[] hostDataFile1;
	delete[] dataOffsetBeg;
	delete[] dataOffsetEnd;
	delete[] doppler;

	return 0;
}