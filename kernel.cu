﻿#ifndef LIBRARIES
#define LIBRARIES
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <chrono>
using namespace std;
#endif

#include"functions.cuh"

int main() {
	hipDeviceReset();//reset device
	
	
	int fftsize = 32768;
	int numOfBytes = fftsize* 31;//bytes to read
	int samplesOfSignal = 4* numOfBytes;//complex samples
	int overlap = 32; //samples of overlaping 
	bool readbinary = 0, writebinary = 1;
	string* fileNames;
	fileNames = new string[2]{ "result.bin", "sin124xN.txt" };//names of files
	
	
	int numofFFTs = samplesOfSignal / (fftsize - overlap);
	int samples= numofFFTs * fftsize;//total samples needed
	if(samplesOfSignal > samples){samples = samplesOfSignal;}
	char *datainBytes, *deviceDatainBytes;
	hipfftComplex *deviceDataFile1, *deviceDataFile2;
	hipfftComplex *hostDataFile1, *hostDataFile2;

	int blockSize = 1024;
	int numBlocks;

	int const iterations = 5;
	long long read_elapsed_secs[iterations];
	long long fft_elapsed_secs[iterations];
	long long mul_elapsed_secs[iterations];
	long long ifft_elapsed_secs[iterations];
	long long write_elapsed_secs[iterations];
	long long elapsed_secs[iterations];
	long long shift_elapsed_secs[iterations];
	
	//ALLOCATE
	datainBytes = (char *)malloc(sizeof(char) * numOfBytes);
	CudaSafeCall(hipMalloc((void **)&deviceDatainBytes, sizeof(char)*numOfBytes));
	hostDataFile1 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * samples);
	hostDataFile2 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * fftsize);
	CudaSafeCall(hipMalloc(&deviceDataFile1, sizeof(hipfftComplex)*samples));
	CudaSafeCall(hipMalloc(&deviceDataFile2, sizeof(hipfftComplex)*fftsize));
	hipDeviceSynchronize();
	
	//FFT&IFFT PLANS
	hipfftHandle plan;
	hipfftHandle planref;
	hipfftHandle inverseplan;
	planfftFunction(fftsize, numofFFTs, overlap, &plan);
	planfftFunction(fftsize, 1, 0, &planref);
	planifftFunction(fftsize, numofFFTs, overlap, &inverseplan);

	//LOOP
	int i;
	for (i = 0; i < iterations; i++) {
		auto Begin = std::chrono::high_resolution_clock::now();

		//READ DATA
		auto readdataBeg = chrono::high_resolution_clock::now();
		//readdata(samplesOfSignal, hostDataFile1, fileNames[0], readbinary);
		readrealdata(numOfBytes, datainBytes, fileNames[0]);
		
		auto elapsed_read = chrono::high_resolution_clock::now() - readdataBeg;
		readdata(fftsize - overlap, hostDataFile2, fileNames[1], readbinary);
		

		//CHECK: READED DATA 
		//cout << "read done\n";
		//writedata(fftsize, hostDataFile1, "rawsin.txt", writebinary);
		//writedata(fftsize, hostDataFile2, "rawsin2.txt", writebinary);

		//MEMORY FROM HOST TO DEVICE
		//CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
		CudaSafeCall(hipMemcpy(deviceDatainBytes, datainBytes, sizeof(char)*numOfBytes, hipMemcpyHostToDevice));
		CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
		hipDeviceSynchronize();

		//MASK AND SHIFT & EXTEND REFERENCE SIGNAL
		auto shiftBeg = chrono::high_resolution_clock::now();
		numBlocks = (numOfBytes + blockSize - 1) / blockSize;
		maskandshift << <numBlocks, blockSize >> > (deviceDatainBytes, deviceDataFile1, numOfBytes);
		CudaCheckError();
		

		numBlocks = (fftsize + blockSize - 1) / blockSize;
		
		extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
		CudaCheckError();
		hipDeviceSynchronize();
		auto elapsed_shift = chrono::high_resolution_clock::now() - shiftBeg;

		//FFT
		auto fftBeg = chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(plan, deviceDataFile1, deviceDataFile1, HIPFFT_FORWARD));
		cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));
		hipDeviceSynchronize();
		auto elapsed_fft = chrono::high_resolution_clock::now() - fftBeg;

		//CHECK: MEMORY FROM DEVICE TO HOST (only for printing fft)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samples, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();

		//CHECK: FFT (only for printing fft)
		//fprintf(stderr, "%d FFt done of elements %d each\n", numofFFTs,fftsize);
		//writedata(fftsize, hostDataFile1, "fft.txt", writebinary);

		//COMPLEX CONJUGATE AND MULTIPLICATION
		numBlocks = (samples + blockSize - 1) / blockSize;
		auto mulBeg = chrono::high_resolution_clock::now();
		multip << <numBlocks, blockSize >> > (samples, deviceDataFile1, deviceDataFile2, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		auto elapsed_mul = chrono::high_resolution_clock::now() - mulBeg;

		//CHECK: MEMORY FROM DEVICE TO HOST (only for printing multiplication result)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samples, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();

		//CHECK: MULTIPLICATION (only for printing multiplication result)
		//cout << "multiplication done\n";
		//writedata(fftsize, hostDataFile1, "mult.txt", writebinary);


		//IFFT (To obtain original again it has to be devided for the # of elements)
		auto ifftBeg = chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(inverseplan, deviceDataFile1, deviceDataFile1, HIPFFT_BACKWARD));
		hipDeviceSynchronize();
		auto elapsed_ifft = chrono::high_resolution_clock::now() - ifftBeg;

		//MEMORY FROM HOST TO DEVICE FOR OUTPUT
		CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		
		//OUTPUT
		//cout << "IFFT done\n";
		auto writeBeg = chrono::high_resolution_clock::now();
		writedata(fftsize-overlap, hostDataFile1, "output.txt", writebinary);

		//ELAPSED TIME
		auto elapsed_write = chrono::high_resolution_clock::now() - writeBeg;
		auto elapsed_total = chrono::high_resolution_clock::now() - Begin;

		read_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_read).count();
		shift_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_shift).count();
		fft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_fft).count();
		mul_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_mul).count();
		ifft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_ifft).count();
		write_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_write).count();
		elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_total).count();
	}

	writetime(iterations, "Times_op3.txt", read_elapsed_secs, shift_elapsed_secs, fft_elapsed_secs,
		mul_elapsed_secs, ifft_elapsed_secs, write_elapsed_secs, elapsed_secs);

	cufftSafeCall(hipfftDestroy(plan));
	cufftSafeCall(hipfftDestroy(planref));
	cufftSafeCall(hipfftDestroy(inverseplan));
	free(hostDataFile1);
	free(hostDataFile2);
	hipFree(deviceDataFile1);
	hipFree(deviceDataFile2);
	hipDeviceReset();
	delete[] fileNames;
	
	return 0;
}