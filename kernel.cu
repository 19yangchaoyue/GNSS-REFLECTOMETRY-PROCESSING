﻿#ifndef LIBRARIES
#define LIBRARIES
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <chrono>
#include <npps.h>
#define PI 3.14159265
using namespace std;
#endif

#include"functions.cuh"

int main() {
	hipDeviceReset();//reset device
	
	
	int fftsize = 32768;
	int numofFFTs = 12;
	int overlap = 32; //samples of overlaping 
	bool readbinary = 1, writebinary = 0;
	int quantofAverageIncoherent = 4;
	int const iterations = 1;
	string* fileNames;
	fileNames = new string[3]{"prn_L1CA_32_100_fd_1e3.bin", "prn_L1CA_32_100.bin","Result.txt"};//names of files
	

	int samplesOfSignal = numofFFTs * (fftsize-overlap);//samples of data
	int samplesWithOverlap= numofFFTs * fftsize;//total samples needed
	if(samplesOfSignal > samplesWithOverlap){ samplesWithOverlap = samplesOfSignal;}
	int inchoerentNumofFFT = numofFFTs/ quantofAverageIncoherent;
	int fSampling = (32 * 1.023 * 1000000);
	int blockSize = 1024;
	int numBlocks, nBufferSize, samplePhaseMantain,i;
	
	
	int *devicearrayPos;
	hipfftComplex *deviceDataFile1, *deviceDataFile2, *hostDataFile1, *hostDataFile2;
	Npp32f *deviceIncoherentSum, *devicearrayMaxs;
	Npp8u * pDeviceBuffer;
	
	long long read_elapsed_secs[iterations], fft_elapsed_secs[iterations], mul_elapsed_secs[iterations], ifft_elapsed_secs[iterations],write_elapsed_secs[iterations], elapsed_secs[iterations], shift_elapsed_secs[iterations];
	
	//ALLOCATE
	int *hostarrayPos = new int[inchoerentNumofFFT];
	Npp32f *hostarrayMaxs = new Npp32f[inchoerentNumofFFT];
	hostDataFile1 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * samplesWithOverlap);
	hostDataFile2 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * fftsize);
	CudaSafeCall(hipMalloc(&deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap));
	CudaSafeCall(hipMalloc(&deviceDataFile2, sizeof(hipfftComplex)*fftsize));
	CudaSafeCall(hipMalloc(&deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize));
	CudaSafeCall(hipMalloc(&devicearrayPos, sizeof(int)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT));
	nppsSumGetBufferSize_32f(fftsize, &nBufferSize);
	CudaSafeCall(hipMalloc((void **)(&pDeviceBuffer), nBufferSize));
	hipDeviceSynchronize();
	
	//FFT&IFFT PLANS
	hipfftHandle plan;
	hipfftHandle planref;
	hipfftHandle inverseplan;
	planfftFunction(fftsize, numofFFTs, overlap, &plan);
	planfftFunction(fftsize, 1, 0, &planref);
	planifftFunction(fftsize, numofFFTs, 0, &inverseplan);

	//LOOP
	
	for (i = 0; i < iterations; i++) {
		auto Begin = std::chrono::high_resolution_clock::now();

		//READ DATA
		auto readdataBeg = chrono::high_resolution_clock::now();
		readdata(samplesOfSignal, hostDataFile1, fileNames[0], readbinary);
		readdata(fftsize - overlap, hostDataFile2, fileNames[1], readbinary);
		auto elapsed_read = chrono::high_resolution_clock::now() - readdataBeg;


		//CHECK: READED DATA 
		//cout << "read done\n";
		//writedata(samplesOfSignal/2, hostDataFile1, "rawsin.txt", writebinary);
		//writedata(fftsize- overlap, hostDataFile2, "rawsin2.txt", writebinary);

		//MEMORY FROM HOST TO DEVICE
		CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
		CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
		hipDeviceSynchronize();

		//MULTIPLY BY DOPPLER
		samplePhaseMantain = (i * fftsize*numofFFTs)%fSampling;
		numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		applyDoppler << <numBlocks, blockSize >> > (samplesOfSignal, deviceDataFile1, -1000, fSampling, samplePhaseMantain);
		CudaCheckError();
		hipDeviceSynchronize();

	
		//CHECK: doppler (only for printing doppler)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesOfSignal/2, hostDataFile1, "dopplerout.txt", writebinary);
		
		//EXTEND REFERENCE SIGNAL
		numBlocks = (fftsize + blockSize - 1) / blockSize;
		auto shiftBeg = chrono::high_resolution_clock::now();
		extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
		CudaCheckError();
		hipDeviceSynchronize();
		auto elapsed_shift = chrono::high_resolution_clock::now() - shiftBeg;

		//FFT
		auto fftBeg = chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(plan, deviceDataFile1, deviceDataFile1, HIPFFT_FORWARD));
		cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));
		hipDeviceSynchronize();
		auto elapsed_fft = chrono::high_resolution_clock::now() - fftBeg;


		//CHECK: FFT (only for printing fft)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samples, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(fftsize, hostDataFile1, "fft.txt", writebinary);

		//COMPLEX CONJUGATE AND MULTIPLICATION
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;
		auto mulBeg = chrono::high_resolution_clock::now();
		multip << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, deviceDataFile2, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		auto elapsed_mul = chrono::high_resolution_clock::now() - mulBeg;

	

		//CHECK: MULTIPLICATION (only for printing multiplication result)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "mult.txt", writebinary);
		
		//IFFT (To obtain original again it has to be devided for the # of elements)
		auto ifftBeg = chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(inverseplan, deviceDataFile1, deviceDataFile1, HIPFFT_BACKWARD));
		hipDeviceSynchronize();
		auto elapsed_ifft = chrono::high_resolution_clock::now() - ifftBeg;
		
		//INCOHERENT SUM
		numBlocks = (inchoerentNumofFFT*fftsize + blockSize - 1) / blockSize;
		inchoerentSum << <numBlocks, blockSize >> > (inchoerentNumofFFT*fftsize, deviceDataFile1, deviceIncoherentSum, quantofAverageIncoherent, fftsize);
		hipDeviceSynchronize();

		//MAXIMUM AND STD
		maxAndStd(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayMaxs, devicearrayPos, pDeviceBuffer);



		//CHECK: IFFT OR incho (not both at the same time)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize, hipMemcpyDeviceToHost)); //TO PRINT INCHO SUM
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost)); //TO PRINT IFFT RESULT
		//hipDeviceSynchronize();
		//writeIncohtxt(inchoerentNumofFFT*fftsize, hostDataFile1, "incoh.txt");//TO PRINT INCHO SUM
		//writedata(samplesWithOverlap, hostDataFile1, fileNames[2], writebinary);//TO PRINT IFFT RESULT 


		//MEMORY FROM HOST TO DEVICE FOR OUTPUT
		CudaSafeCall(hipMemcpy(hostarrayMaxs, devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayPos, devicearrayPos, sizeof(int)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		
		//OUTPUT
		//cout<< hostDataFile1[0].x << " incho\n";

		auto writeBeg = chrono::high_resolution_clock::now();

		
		
		writeMaxstxt(inchoerentNumofFFT, hostarrayMaxs, hostarrayPos, "Maximums.txt");

		//ELAPSED TIME
		auto elapsed_write = chrono::high_resolution_clock::now() - writeBeg;
		auto elapsed_total = chrono::high_resolution_clock::now() - Begin;

		read_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_read).count();
		shift_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_shift).count();
		fft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_fft).count();
		mul_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_mul).count();
		ifft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_ifft).count();
		write_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_write).count();
		elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_total).count();
	}

	writetime(iterations, "Times_op3.txt", read_elapsed_secs, shift_elapsed_secs, fft_elapsed_secs,
		mul_elapsed_secs, ifft_elapsed_secs, write_elapsed_secs, elapsed_secs);

	//FREE MEMORY
	cufftSafeCall(hipfftDestroy(plan));
	cufftSafeCall(hipfftDestroy(planref));
	cufftSafeCall(hipfftDestroy(inverseplan));
	hipFree(deviceDataFile1);
	hipFree(deviceDataFile2);
	hipFree(deviceIncoherentSum);
	hipFree(devicearrayPos);
	hipFree(devicearrayMaxs);
	hipFree(pDeviceBuffer);
	hipDeviceReset();
	delete[] fileNames;
	delete[] hostarrayPos;
	delete[] hostarrayMaxs;
	delete[] hostDataFile2;
	delete[] hostDataFile1;


	return 0;
}