﻿#ifndef LIBRARIES
#define LIBRARIES
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <chrono>
#include <npps.h>
#define PI 3.14159265
using namespace std;
#endif

#include"functions.cuh"

int main(int argc, const char* argv[]) {
	hipDeviceReset();//reset device
	
	//READ CONFIG FILE
	int fftsize, fSampling, numofFFTs, overlap, quantofAverageIncoherent, blockSize, peakRangeStd, peakSamplesToSave;
	int const numofDataLines = atoi(argv[2]);//substitut d'iterations
	string *fileDataNames, *fileRefNames;
	int *dataOffsetBeg, *dataOffsetEnd;
	int *doppler;

	fileDataNames = new string[numofDataLines];
	fileRefNames = new string[numofDataLines];
	dataOffsetBeg = new int[numofDataLines];
	dataOffsetEnd = new int[numofDataLines];
	doppler = new int[numofDataLines];

	readConfig(argv[1], numofDataLines, &fftsize, &numofFFTs, &overlap, &fSampling, &blockSize, &peakRangeStd, &peakSamplesToSave, &quantofAverageIncoherent, dataOffsetBeg, dataOffsetEnd, doppler, fileDataNames, fileRefNames);
	checkInputConfig(argc, argv, numofDataLines, fftsize, numofFFTs, overlap, fSampling, blockSize, peakRangeStd, peakSamplesToSave, quantofAverageIncoherent, dataOffsetBeg, dataOffsetEnd, doppler, fileDataNames, fileRefNames);

	//OTHER DECLARATIONS
	int samplesOfSignal = (numofFFTs * (fftsize-overlap))+overlap;//samples of complex data
	int bytesToRead = samplesOfSignal/4;
	if (samplesOfSignal % 4 != 0) { cout << "Warning bytesToRead rounded toward negative infinity: samplesOfSignal%4!=0 \n "; }
	int samplesWithOverlap= numofFFTs * fftsize;//total samples needed
	if(samplesOfSignal > samplesWithOverlap){ samplesWithOverlap = samplesOfSignal;}
	int inchoerentNumofFFT = numofFFTs/ quantofAverageIncoherent;
	if (numofFFTs % quantofAverageIncoherent != 0) {
		cout << "Error: numofFFTs / quantofAverageIncoherent != 0\n ";
		exit(-1);
	}

	string outputName;
	int numBlocks, nBufferSize,i;
	unsigned long long samplePhaseMantain;

	char *hostBytesOfData, *deviceBytesOfData;
	int *devicearrayPos,*hostarrayPos;
	hipfftComplex *deviceDataFile1, *deviceDataFile2, *hostDataFile1, *hostDataFile2, *deviceDataToSave;
	Npp32f *deviceIncoherentSum, *devicearrayMaxs, *devicearrayStd, *hostarrayMaxs, *hostarrayStd;
	Npp8u * pDeviceBuffer;
	
	long long *read_elapsed_secs,*write_elapsed_secs, *elapsed_secs;
	
	//ALLOCATE
	read_elapsed_secs = new long long[numofDataLines];
	write_elapsed_secs = new long long[numofDataLines];
	elapsed_secs = new long long[numofDataLines];
	hostBytesOfData = (char *)malloc(sizeof(char) * bytesToRead);
	hostarrayPos = new int[inchoerentNumofFFT];
	hostarrayMaxs = new Npp32f[inchoerentNumofFFT];
	hostarrayStd = new Npp32f[inchoerentNumofFFT];
	hostDataFile1 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * samplesWithOverlap);
	hostDataFile2 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * fftsize);
	CudaSafeCall(hipMalloc(&deviceBytesOfData, sizeof(char)*bytesToRead));
	CudaSafeCall(hipMalloc(&deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap));
	CudaSafeCall(hipMalloc(&deviceDataToSave, sizeof(hipfftComplex)*peakSamplesToSave*numofFFTs));
	CudaSafeCall(hipMalloc(&deviceDataFile2, sizeof(hipfftComplex)*fftsize));
	CudaSafeCall(hipMalloc(&deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize));
	CudaSafeCall(hipMalloc(&devicearrayPos, sizeof(int)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT));
	nppsSumGetBufferSize_32f(fftsize, &nBufferSize);
	CudaSafeCall(hipMalloc((void **)(&pDeviceBuffer), nBufferSize));
	hipDeviceSynchronize();
	
	//FFT&IFFT PLANS
	hipfftHandle plan;
	hipfftHandle planref;
	hipfftHandle inverseplan;
	planfftFunction(fftsize, numofFFTs, overlap, &plan);
	planfftFunction(fftsize, 1, 0, &planref);
	planifftFunction(fftsize, numofFFTs, 0, &inverseplan);
	hipDeviceSynchronize();

	//LOOP
	for (i = 0; i < numofDataLines; i++) {
		
		auto Begin = std::chrono::high_resolution_clock::now();
		//READ DATA
		readdata(dataOffsetEnd[i]-dataOffsetBeg[i], dataOffsetBeg[i], hostDataFile1, fileDataNames[i]);
		readdata(fftsize - overlap,0, hostDataFile2, fileRefNames[i]);

		/*if (dataOffsetEnd[i] - dataOffsetBeg[i] > bytesToRead) { cout << "Indices of reading in config file exceed bytesToRead decleared"; }
		if ((dataOffsetEnd[i] - dataOffsetBeg[i])%(fftsize*quantofAverageIncoherent)!=0) {
		cout << "Warning length of data won't complete last incho sum in DATALINE: "<<i<<"\n"; }
		readRealData(dataOffsetEnd[i] - dataOffsetBeg[i], dataOffsetBeg[i],bytesToRead, hostBytesOfData, fileNames[i]);*/
		auto elapsed_read = chrono::high_resolution_clock::now() - Begin;

		CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
		CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
		/*CudaSafeCall(hipMemcpy(deviceBytesOfData, hostBytesOfData, sizeof(char)*bytesToRead, hipMemcpyHostToDevice));*/
		hipDeviceSynchronize();
		
		//CHECK: READED DATA 
		//writedata(samplesOfSignal/2, hostDataFile1, "rawsin.txt");
		//writedata(fftsize- overlap, hostDataFile2, "rawsin2.txt");

		//MASK AND SHIFT
		/*numBlocks = (bytesToRead + blockSize - 1) / blockSize;
		maskAndShift << <numBlocks, blockSize >> > (deviceBytesOfData, deviceDataFile1, bytesToRead);
		CudaCheckError();
		hipDeviceSynchronize();
		*/
		//MULTIPLY BY DOPPLER
		samplePhaseMantain = (i * fftsize*numofFFTs);// %fSampling;----
		numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		applyDoppler << <numBlocks, blockSize >> > (samplesOfSignal, deviceDataFile1, doppler[i], fSampling, samplePhaseMantain);
		CudaCheckError();
		hipDeviceSynchronize();
	
		//CHECK: doppler (only for printing doppler)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesOfSignal/2, hostDataFile1, "dopplerout.txt");
		
		//EXTEND REFERENCE SIGNAL
		numBlocks = (fftsize + blockSize - 1) / blockSize;
		extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
		CudaCheckError();
		hipDeviceSynchronize();

		//FFT
		cufftSafeCall(hipfftExecC2C(plan, deviceDataFile1, deviceDataFile1, HIPFFT_FORWARD));
		cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));
		hipDeviceSynchronize();

		//CHECK: FFT (only for printing fft)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "fft.txt");

		//COMPLEX CONJUGATE AND MULTIPLICATION
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;
		multip << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, deviceDataFile2, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();

		//CHECK: MULTIPLICATION (only for printing multiplication result)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "mult.txt");
		
		//IFFT
		cufftSafeCall(hipfftExecC2C(inverseplan, deviceDataFile1, deviceDataFile1, HIPFFT_BACKWARD));
		hipDeviceSynchronize();
		
		//SCALE (To take back original signal it has to be devided for the fftsize)
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;		
		scale << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		
		//CHECK: IFFT 
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost)); 
		//hipDeviceSynchronize();	
		//writedata(samplesWithOverlap, hostDataFile1,  "IFFT-result.bin");

		//INCOHERENT SUM
		numBlocks = (inchoerentNumofFFT*fftsize + blockSize - 1) / blockSize;
		inchoerentSum << <numBlocks, blockSize >> > (inchoerentNumofFFT*fftsize, deviceDataFile1, deviceIncoherentSum, quantofAverageIncoherent, fftsize);
		CudaCheckError(); 
		hipDeviceSynchronize();

		//CHECK: INCOHERENT
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writeIncoh(inchoerentNumofFFT*fftsize, hostDataFile1, "incoh.bin");
	
		//MAXIMUM
		maxCompute(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayMaxs, devicearrayPos, pDeviceBuffer);
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostarrayPos, devicearrayPos, sizeof(int)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayMaxs, devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		
		//SAVE PEAKS
		numBlocks = (numofFFTs*peakSamplesToSave + blockSize - 1) / blockSize;
		savePeak << <numBlocks, blockSize >> > (numofFFTs, deviceDataFile1, deviceDataToSave, peakSamplesToSave, quantofAverageIncoherent, fftsize, devicearrayPos);
		CudaCheckError();
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataToSave, sizeof(hipComplex)*numofFFTs*peakSamplesToSave, hipMemcpyDeviceToHost));
		
		//STD
		stdCompute(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayStd, hostarrayPos, pDeviceBuffer, peakRangeStd);
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostarrayStd, devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		
		//OUTPUT
		auto writeBeg = chrono::high_resolution_clock::now();
		writeMaxs(inchoerentNumofFFT, hostarrayMaxs, hostarrayPos, hostarrayStd, "results/Maximums.txt");
		outputName = "results/PeaksIteration"+ to_string(i);
		outputName = outputName + ".bin";
		cout << outputName << "\n";
		writedata(numofFFTs*peakSamplesToSave, hostDataFile1, outputName);
	
		//ELAPSED TIME
		auto elapsed_write = chrono::high_resolution_clock::now() - writeBeg;
		auto elapsed_total = chrono::high_resolution_clock::now() - Begin;

		read_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_read).count();
		write_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_write).count();
		elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_total).count();
	}

	writetime(numofDataLines, "results/Times_op3.txt", read_elapsed_secs, write_elapsed_secs, elapsed_secs);

	//FREE MEMORY
	cufftSafeCall(hipfftDestroy(plan));
	cufftSafeCall(hipfftDestroy(planref));
	cufftSafeCall(hipfftDestroy(inverseplan));
	hipFree(deviceDataFile1);
	hipFree(deviceDataFile2);
	hipFree(deviceIncoherentSum);
	hipFree(devicearrayPos);
	hipFree(deviceBytesOfData);
	hipFree(devicearrayMaxs);
	hipFree(deviceDataToSave);
	hipFree(pDeviceBuffer);
	hipFree(devicearrayStd);
	hipDeviceReset();
	delete[] fileDataNames;
	delete[] fileRefNames;
	delete[] hostBytesOfData;
	delete[] hostarrayPos;
	delete[] hostarrayMaxs;
	delete[] hostarrayStd;
	delete[] hostDataFile2;
	delete[] hostDataFile1;
	delete[] dataOffsetBeg;
	delete[] dataOffsetEnd;
	delete[] doppler;
	delete[] read_elapsed_secs;
	delete[] write_elapsed_secs;
	delete[] elapsed_secs;
	
	return 0;
}