﻿#ifndef LIBRARIES
#define LIBRARIES
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <chrono>
#include <npps.h>
#define PI 3.14159265
using namespace std;
#endif

#include"functions.cuh"

int main(int argc, const char* argv[]) {
	hipDeviceReset();//reset device
	
	//READ CONFIG FILE
	int fftsize, fSampling, numofFFTs, overlap, quantofAverageIncoherent, blockSize, peakRangeStd, peakSamplesToSave,ddmspan,ddmnumdiv;
	int const numofDataLines = atoi(argv[2]);//substitut d'iterations
	string *fileDataNames, fileRefName;
	int *dataOffsetBeg, *dataOffsetEnd;
	int *doppler;

	fileDataNames = new string[numofDataLines];
	dataOffsetBeg = new int[numofDataLines];
	dataOffsetEnd = new int[numofDataLines];
	doppler = new int[numofDataLines];

	readConfig(argv[1], numofDataLines, &fftsize, &numofFFTs, &overlap, &fSampling, &blockSize, &peakRangeStd, &peakSamplesToSave, &quantofAverageIncoherent, dataOffsetBeg, dataOffsetEnd, doppler, fileDataNames, &fileRefName, &ddmspan, &ddmnumdiv);
	checkInputConfig(argc, argv, numofDataLines, fftsize, numofFFTs, overlap, fSampling, blockSize, peakRangeStd, peakSamplesToSave, quantofAverageIncoherent, dataOffsetBeg, dataOffsetEnd, doppler, fileDataNames, fileRefName, ddmspan, ddmnumdiv);

	//OTHER DECLARATIONS
	int samplesOfSignal = (numofFFTs * (fftsize-overlap))+overlap;//samples of complex data
	int bytesToRead = samplesOfSignal/4;
	if (samplesOfSignal % 4 != 0) { cout << "Warning bytesToRead rounded toward negative infinity: samplesOfSignal%4!=0 \n "; }
	int samplesWithOverlap= numofFFTs * fftsize;//total samples needed
	if(samplesOfSignal > samplesWithOverlap){ samplesWithOverlap = samplesOfSignal;}
	int inchoerentNumofFFT = numofFFTs/ quantofAverageIncoherent;
	if (numofFFTs % quantofAverageIncoherent != 0) {
		cout << "Error: numofFFTs / quantofAverageIncoherent != 0\n ";
		exit(-1);
	}

	string outputName;
	int numBlocks, nBufferSize,i;
	unsigned long long samplePhaseMantain;

	char *hostBytesOfData, *deviceBytesOfData;
	int *devicearrayPos,*hostarrayPos;
	hipfftComplex *deviceDataFile1, *deviceDataFile2, *hostDataFile1, *hostDataFile2, *deviceDataToSave;
	Npp32f *deviceIncoherentSum, *devicearrayMaxs, *devicearrayStd, *hostarrayMaxs, *hostarrayStd;
	Npp8u * pDeviceBuffer;
	
	long long *read_elapsed_secs,*write_elapsed_secs, *elapsed_secs, *mask_elapsed_secs, *doppler_elapsed_secs, 
		 *fft_elapsed_secs, *mult_elapsed_secs,*ifft_elapsed_secs, *scale_elapsed_secs, *incho_elapsed_secs
		, *max_elapsed_secs, *savep_elapsed_secs, *std_elapsed_secs;
	
	//ALLOCATE
	read_elapsed_secs = new long long[numofDataLines];
	mask_elapsed_secs = new long long[numofDataLines];
	doppler_elapsed_secs = new long long[numofDataLines];
	fft_elapsed_secs = new long long[numofDataLines];
	mult_elapsed_secs = new long long[numofDataLines];
	ifft_elapsed_secs = new long long[numofDataLines];
	scale_elapsed_secs = new long long[numofDataLines];
	incho_elapsed_secs = new long long[numofDataLines];
	max_elapsed_secs = new long long[numofDataLines];
	savep_elapsed_secs = new long long[numofDataLines];
	std_elapsed_secs = new long long[numofDataLines];
	write_elapsed_secs = new long long[numofDataLines];
	elapsed_secs = new long long[numofDataLines];

	hostBytesOfData = (char *)malloc(sizeof(char) * bytesToRead);
	hostarrayPos = new int[inchoerentNumofFFT];
	hostarrayMaxs = new Npp32f[inchoerentNumofFFT];
	hostarrayStd = new Npp32f[inchoerentNumofFFT];
	hostDataFile1 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * samplesWithOverlap);
	hostDataFile2 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * fftsize);
	CudaSafeCall(hipMalloc(&deviceBytesOfData, sizeof(char)*bytesToRead));
	CudaSafeCall(hipMalloc(&deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap));
	CudaSafeCall(hipMalloc(&deviceDataToSave, sizeof(hipfftComplex)*peakSamplesToSave*numofFFTs));
	CudaSafeCall(hipMalloc(&deviceDataFile2, sizeof(hipfftComplex)*fftsize));
	CudaSafeCall(hipMalloc(&deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize));
	CudaSafeCall(hipMalloc(&devicearrayPos, sizeof(int)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT));
	nppsSumGetBufferSize_32f(fftsize, &nBufferSize);
	CudaSafeCall(hipMalloc((void **)(&pDeviceBuffer), nBufferSize));
	hipDeviceSynchronize();
	
	cout << "GPU mem used: " << sizeof(char)*bytesToRead + sizeof(hipfftComplex)*samplesWithOverlap + 
		sizeof(hipfftComplex)*peakSamplesToSave*numofFFTs + sizeof(hipfftComplex)*fftsize + sizeof(Npp32f)*inchoerentNumofFFT*fftsize
		+ sizeof(int)*inchoerentNumofFFT + sizeof(Npp32f)*inchoerentNumofFFT + sizeof(Npp32f)*inchoerentNumofFFT<<" bytes\n";
	

	//FFT&IFFT PLANS
	hipfftHandle plan;
	hipfftHandle planref;
	hipfftHandle inverseplan;
	planfftFunction(fftsize, numofFFTs, overlap, &plan);
	planfftFunction(fftsize, 1, 0, &planref);
	planifftFunction(fftsize, numofFFTs, 0, &inverseplan);
	hipDeviceSynchronize();

	//READ, EXTEND AND FFT OF REF SIGNAL
	readdata(fftsize - overlap, 0, hostDataFile2, fileRefName);

	//writedata(fftsize - overlap, hostDataFile2, "rawref2.bin");

	CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	
	numBlocks = (fftsize + blockSize - 1) / blockSize;
	extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
	CudaCheckError();
	hipDeviceSynchronize();

	cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));


	//LOOP
	for (i = 0; i < numofDataLines; i++) {
		
		auto begin = std::chrono::high_resolution_clock::now();
		//READ DATA
		/*readdata(dataOffsetEnd[i]-dataOffsetBeg[i], dataOffsetBeg[i], hostDataFile1, fileDataNames[i]);*/
		readRealData(dataOffsetEnd[i] - dataOffsetBeg[i], dataOffsetBeg[i],bytesToRead, hostBytesOfData, fileDataNames[i]);
		
		/*CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));*/
		CudaSafeCall(hipMemcpy(deviceBytesOfData, hostBytesOfData, sizeof(char)*bytesToRead, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
		auto elapsed_read = chrono::high_resolution_clock::now() - begin;

		//MASK AND SHIFT
		auto maskbeg = std::chrono::high_resolution_clock::now();
		numBlocks = (bytesToRead + blockSize - 1) / blockSize;
		maskAndShift << <numBlocks, blockSize >> > (deviceBytesOfData, deviceDataFile1, bytesToRead);
		CudaCheckError();
		hipDeviceSynchronize();
		auto mask_elapsed = chrono::high_resolution_clock::now() - maskbeg;
		
		//CHECK: RAW DATA 
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*(dataOffsetEnd[i] - dataOffsetBeg[i])*4, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata((dataOffsetEnd[i] - dataOffsetBeg[i])*4, hostDataFile1, "rawdata.bin");

		//MULTIPLY BY DOPPLER
		auto dopplerbeg = std::chrono::high_resolution_clock::now();
		samplePhaseMantain = (i * fftsize*numofFFTs);// %fSampling;----
		numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		applyDoppler << <numBlocks, blockSize >> > (samplesOfSignal, deviceDataFile1, doppler[i], fSampling, samplePhaseMantain);
		CudaCheckError();
		hipDeviceSynchronize();
		auto doppler_elapsed = chrono::high_resolution_clock::now() - dopplerbeg;
		//CHECK: doppler (only for printing doppler)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesOfSignal/2, hostDataFile1, "dopplerout.txt");

		//FFT
		auto fftbeg = std::chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(plan, deviceDataFile1, deviceDataFile1, HIPFFT_FORWARD));
		hipDeviceSynchronize();
		auto fft_elapsed = chrono::high_resolution_clock::now() - fftbeg;

		//CHECK: FFT (only for printing fft)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "fft.txt");

		//COMPLEX CONJUGATE AND MULTIPLICATION
		auto multbeg = std::chrono::high_resolution_clock::now();
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;
		multip << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, deviceDataFile2, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		auto mult_elapsed = chrono::high_resolution_clock::now() - multbeg;

		//CHECK: MULTIPLICATION (only for printing multiplication result)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "mult.txt");
		
		//IFFT
		auto ifftbeg = std::chrono::high_resolution_clock::now();
		cufftSafeCall(hipfftExecC2C(inverseplan, deviceDataFile1, deviceDataFile1, HIPFFT_BACKWARD));
		hipDeviceSynchronize();
		auto ifft_elapsed = chrono::high_resolution_clock::now() - ifftbeg;

		//SCALE (To take back original signal it has to be devided for the fftsize)
		auto scalebeg = std::chrono::high_resolution_clock::now();
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;		
		scale << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		auto scale_elapsed = chrono::high_resolution_clock::now() - scalebeg;

		//CHECK: IFFT 
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost)); 
		//hipDeviceSynchronize();	
		//writedata(samplesWithOverlap, hostDataFile1,  "IFFT-result.bin");

		//INCOHERENT SUM
		auto incohbeg = std::chrono::high_resolution_clock::now();
		numBlocks = (inchoerentNumofFFT*fftsize + blockSize - 1) / blockSize;
		inchoerentSum << <numBlocks, blockSize >> > (inchoerentNumofFFT*fftsize, deviceDataFile1, deviceIncoherentSum, quantofAverageIncoherent, fftsize);
		CudaCheckError(); 
		hipDeviceSynchronize();
		auto incho_elapsed = chrono::high_resolution_clock::now() - incohbeg;
		//CHECK: INCOHERENT
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writeIncoh(inchoerentNumofFFT*fftsize, hostDataFile1, "incoh.bin");
	
		//MAXIMUM
		auto maxbeg = std::chrono::high_resolution_clock::now();
		maxCompute(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayMaxs, devicearrayPos, pDeviceBuffer);
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostarrayPos, devicearrayPos, sizeof(int)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayMaxs, devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		auto max_elapsed = chrono::high_resolution_clock::now() - maxbeg;
		
		//SAVE PEAKS
		auto savepbeg = std::chrono::high_resolution_clock::now();
		numBlocks = (numofFFTs*peakSamplesToSave + blockSize - 1) / blockSize;
		savePeak << <numBlocks, blockSize >> > (numofFFTs, deviceDataFile1, deviceDataToSave, peakSamplesToSave, quantofAverageIncoherent, fftsize, devicearrayPos);
		CudaCheckError();
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataToSave, sizeof(hipComplex)*numofFFTs*peakSamplesToSave, hipMemcpyDeviceToHost));
		auto savep_elapsed = chrono::high_resolution_clock::now() - savepbeg;

		//STD
		auto stdbeg = std::chrono::high_resolution_clock::now();
		stdCompute(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayStd, hostarrayPos, pDeviceBuffer, peakRangeStd);
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostarrayStd, devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		auto std_elapsed = chrono::high_resolution_clock::now() - stdbeg;

		//OUTPUT
		auto writeBeg = chrono::high_resolution_clock::now();
		writeMaxs(inchoerentNumofFFT, hostarrayMaxs, hostarrayPos, hostarrayStd, doppler[i],"results/Maximums.txt");
		outputName = "results/PeaksIteration"+ to_string(i);
		outputName = outputName + ".bin";
		cout << outputName << "\n";
		writedata(numofFFTs*peakSamplesToSave, hostDataFile1, outputName);
	
		//ELAPSED TIME
		auto elapsed_write = chrono::high_resolution_clock::now() - writeBeg;
		auto elapsed_total = chrono::high_resolution_clock::now() - begin;

		read_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_read).count();
		write_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_write).count();
		elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_total).count();
		mask_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(mask_elapsed).count();
		doppler_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(doppler_elapsed).count();
		fft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(fft_elapsed).count();
		mult_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(mult_elapsed).count();
		ifft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(ifft_elapsed).count();
		scale_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(scale_elapsed).count();
		incho_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(incho_elapsed).count();
		max_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(max_elapsed).count();
		savep_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(savep_elapsed).count();
		std_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(std_elapsed).count();
	}

	writetime(numofDataLines, "results/Times.txt", read_elapsed_secs, write_elapsed_secs, elapsed_secs,
		mask_elapsed_secs, doppler_elapsed_secs,
		fft_elapsed_secs, mult_elapsed_secs, ifft_elapsed_secs, scale_elapsed_secs, incho_elapsed_secs
		, max_elapsed_secs, savep_elapsed_secs, std_elapsed_secs);

	//FREE MEMORY
	cufftSafeCall(hipfftDestroy(plan));
	cufftSafeCall(hipfftDestroy(planref));
	cufftSafeCall(hipfftDestroy(inverseplan));
	hipFree(deviceDataFile1);
	hipFree(deviceDataFile2);
	hipFree(deviceIncoherentSum);
	hipFree(devicearrayPos);
	hipFree(deviceBytesOfData);
	hipFree(devicearrayMaxs);
	hipFree(deviceDataToSave);
	hipFree(pDeviceBuffer);
	hipFree(devicearrayStd);
	hipDeviceReset();
	delete[] fileDataNames;
	delete[] hostBytesOfData;
	delete[] hostarrayPos;
	delete[] hostarrayMaxs;
	delete[] hostarrayStd;
	delete[] hostDataFile2;
	delete[] hostDataFile1;
	delete[] dataOffsetBeg;
	delete[] dataOffsetEnd;
	delete[] doppler;
	delete[] read_elapsed_secs;
	delete[] write_elapsed_secs;
	delete[] elapsed_secs;
	delete[] mask_elapsed_secs;
	delete[] doppler_elapsed_secs ;
	delete[] mult_elapsed_secs ;
	delete[] fft_elapsed_secs;
	delete[] ifft_elapsed_secs;
	delete[] scale_elapsed_secs;
	delete[] incho_elapsed_secs ;
	delete[] max_elapsed_secs ;
	delete[] savep_elapsed_secs ;
	delete[] std_elapsed_secs;
	return 0;
}