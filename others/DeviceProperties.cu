#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;



int main() {
	//-------
	//DEVICE MANAGEMENT

	hipDeviceReset();
	int count;
	hipGetDeviceCount(&count);
	fprintf(stderr, "count devices: %i \n", count);
	for (int i = 0; i < count; i++) {
		hipDeviceProp_t aa;
		hipSetDevice(i);
		hipGetDeviceProperties(&aa, i);

		
		
		fprintf(stderr, "Device %i ", i);
		fprintf(stderr, ":\n  Name: %s \n", aa.name);
		fprintf(stderr, "  maxThreadsPerBlock: %i \n", aa.maxThreadsPerBlock);
		fprintf(stderr, "  max dim of block of x: %i \n", aa.maxThreadsDim[0]);
		fprintf(stderr, "  max dim of block of y: %i \n", aa.maxThreadsDim[1]);
		
		
		size_t freeMem, totalMem;

		hipMemGetInfo(&freeMem, &totalMem);

		fprintf(stderr, "  Memory: \n");
		fprintf(stderr, "   Free = %zu, Total = %zu\n", freeMem, totalMem);
	}
	
}