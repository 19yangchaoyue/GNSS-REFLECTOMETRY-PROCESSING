#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;



int main() {
	//-------
	//DEVICE MANAGEMENT

	hipDeviceReset();
	int count;
	hipGetDeviceCount(&count);
	fprintf(stderr, "count devices: %i \n", count);
	for (int i = 0; i < count; i++) {
		hipDeviceProp_t aa;
		hipSetDevice(i);
		hipGetDeviceProperties(&aa, i);

		
		
		fprintf(stderr, "Device %i ", i);
		fprintf(stderr, ":\n  Name: %s \n", aa.name);
		fprintf(stderr, "  maxThreadsPerBlock: %i \n", aa.maxThreadsPerBlock);
		
		size_t freeMem, totalMem;

		hipMemGetInfo(&freeMem, &totalMem);

		fprintf(stderr, "  Memory: \n");
		fprintf(stderr, "   Free = %zu, Total = %zu\n", freeMem, totalMem);
	}
	
}