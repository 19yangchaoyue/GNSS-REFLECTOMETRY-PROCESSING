﻿#include"Hostfunc.cuh"
#include "TextParser.cuh"


//INPUT CONFIG PARSER FUNCTIONS

void readConfig(const char *configFileName, int numofDataLines, int *fftsize, int *numofFFts, int *overlap, int *fSampling, int *blockSize, int *peakRangeStd, int *peakSamplesToSave,
	int* quantOfAverIncoh, int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames,string *fileRefNames, int *ddmRes, int *ddmQuant,bool *interfer
,int *dataOffsetBegInterferometric,int *samplesAvoidMaxs) {

	TextParser t(configFileName);

	TextParserSafeCall(t.seek("*FFTSIZE"));
	*fftsize = t.getint();
	TextParserSafeCall(t.seek("*NUMOFFFTS"));
	*numofFFts = t.getint();
	TextParserSafeCall(t.seek("*QUANTINCOHAVER"));
	*quantOfAverIncoh = t.getint();
	TextParserSafeCall(t.seek("*OVERLAP"));
	*overlap = t.getint();
	TextParserSafeCall(t.seek("*FSAMPLING"));
	*fSampling = t.getint();
	TextParserSafeCall(t.seek("*BLOCKSIZE"));
	*blockSize = t.getint();
	TextParserSafeCall(t.seek("*INTERFEROMETIC"));
	*interfer = t.getint();
	TextParserSafeCall(t.seek("*SAMPLESAVOIDMAX"));
	*samplesAvoidMaxs = t.getint();
	TextParserSafeCall(t.seek("*PEAKRANGESTD"));
	*peakRangeStd = t.getint();
	TextParserSafeCall(t.seek("*PEAKSAMPLESTOSAVE"));
	*peakSamplesToSave = t.getint();
	if (*interfer == false) {
		TextParserSafeCall(t.seek("*REFFILENAME"));
		fileRefNames[0] = t.getword(); 
	}
	TextParserSafeCall(t.seek("*DDMFREQRES"));
	*ddmRes = t.getint();
	TextParserSafeCall(t.seek("*DDMNUMQUANT"));
	*ddmQuant = t.getint();

	if (*ddmQuant % 2 != 1) {
		cout << "ERROR: DDM QUANT has to be odd: 1(original)+2n(symethric)\n";
		exit(1);

	}


	TextParserSafeCall(t.seek("*QUANTDATALINES"));
	if (t.getint() != numofDataLines) {
		cout << "diferent num of Data lines in the file than declared on arguments \n  To execute enter arguments: NameconfigFile.ASE NumofDataLines\n";
		exit(1);
	}

	for (int i = 0; i < numofDataLines; i++) {
		TextParserSafeCall(t.seek("*DATALINE"));

		fileNames[i] = t.getword();
		dataOffsetBeg[i] = t.getint();
		dataOffsetEnd[i] = t.getint();
		doppler[i] = t.getint();
		if (*interfer == true) {
			fileRefNames[i] = t.getword();
			dataOffsetBegInterferometric[i]= t.getint();
		}
	}
}

void checkInputConfig(int argc, const char **argv, int numofDataLines, int fftsize, int numofFFts, int overlap, int fSampling,  int blockSize, int peakRangeStd, int peakSamplesToSave,
	int quantOfAverIncoh,  int *dataOffsetBeg, int *dataOffsetEnd, int *doppler, string *fileNames, string *fileRefNames, int ddmRes, int ddmQuant, bool interfer
, int *dataOffsetBegInterferometric, int samplesAvoidMaxs) {

	if (argc != 3) {
		cout << "Error: Wrong number of arguments\n"; 
		exit(0);
	}

	cout << "\n" << "-ARGS: " << argc << "\n";
	cout << "First: " << argv[0] << "\n";
	cout << "Second: " << argv[1] << "\n";
	cout << "Third: " << argv[2] << "\n\n";

	cout << "-INPUTS:\n";
	cout << "FFT Size: " << fftsize << "\n";
	cout << "Num. of FFT: " << numofFFts << "\n";
	cout << "Overlap: " << overlap << "\n";
	cout << "FSampling: " << fSampling << "\n";
	cout << "Quant of averg Inch.: " << quantOfAverIncoh << "\n";
	cout << "Blok Size: " << blockSize << "\n";
	cout << "Interferometric: " << interfer << "\n";
	cout << "Samples avoid MAxs: " << samplesAvoidMaxs << "\n";
	cout << "Peak samples for the std: " << peakRangeStd << "\n";
	cout << "Peak samples to save: " << peakSamplesToSave << "\n";
	if (interfer == false) {
		cout << "Ref File Name: " << fileRefNames[0] << "\n";
	}
	cout << "DDM Res: " << ddmRes << "\n";
	cout << "DDM Quant: " << ddmQuant << "\n";


	cout << "Num of data lines: " << numofDataLines << "\n";
	cout << "Data lines: \n";
	for (int i = 0; i < numofDataLines; i++) {
		cout << fileNames[i] << "  ";
		cout << dataOffsetBeg[i] << "  ";
		cout << dataOffsetEnd[i] << "  ";
		cout << doppler[i] << "  ";
		if (interfer == true) {
			cout << fileRefNames[0] << "  ";
			cout << dataOffsetBegInterferometric << "\n";
		}
		else {
			cout << "\n";
		}
	}

}

//PREPARE DATA FUNCTIONS

void prepareReference(int fftsize, int overlap,int blockSize ,hipfftComplex *hostDataFile2, hipfftComplex *deviceDataFile2,string fileRefName) {
	
	readdata(fftsize - overlap, 0, hostDataFile2, fileRefName);
	hipfftHandle planref;
	CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	if (overlap > 0) {
		int numBlocks = (fftsize + blockSize - 1) / blockSize;
		extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
		CudaCheckError();
	}
	planfftFunction(fftsize, 1, 0, &planref);
	hipDeviceSynchronize();
	cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));
	hipDeviceSynchronize();
	cufftSafeCall(hipfftDestroy(planref));

}


void prepareData( int dataOffsetEnd,int dataOffsetBeg, int bytesToRead, char *hostBytesOfData, string fileDataNames,
	char *deviceBytesOfData, int blockSize, int ddmQuant, int samplesOfSignal, int samplesWithOverlap, hipfftComplex *deviceDataFile1
     ,int numofFFTs, int fftsize, hipfftComplex *hostDataFile1, chrono::nanoseconds *elapsed_read, chrono::nanoseconds *mask_elapsed
	,chrono::nanoseconds *extenddop_elapsed) {
	
	auto begin = std::chrono::high_resolution_clock::now();
	//READ DATA
	//readdata(dataOffsetEnd[i]-dataOffsetBeg[i], dataOffsetBeg[i], hostDataFile1, fileDataNames[i]);
	readRealData(dataOffsetEnd - dataOffsetBeg, dataOffsetBeg, bytesToRead, hostBytesOfData, fileDataNames);

	//CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(deviceBytesOfData, hostBytesOfData, sizeof(char)*bytesToRead, hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	*elapsed_read = chrono::high_resolution_clock::now() - (begin);

	//MASK AND SHIFT
	auto maskbeg = std::chrono::high_resolution_clock::now();
	int numBlocks = (bytesToRead + blockSize - 1) / blockSize;
	maskAndShift << <numBlocks, blockSize >> > (deviceBytesOfData, deviceDataFile1, bytesToRead);
	CudaCheckError();
	hipDeviceSynchronize();
	*mask_elapsed = chrono::high_resolution_clock::now() - maskbeg;

	//EXTEND FOR DOPPLER
	auto extenddopbeg = std::chrono::high_resolution_clock::now();
	if (ddmQuant > 1) {
		numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		extendRefSignal << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, numofFFTs * fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
	}
	*extenddop_elapsed = chrono::high_resolution_clock::now() - extenddopbeg;


}

//FFT PLANS FUNCTIONS

void planfftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize - overlap, odist = fftsize;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void planifftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {
	
	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

size_t planMemEstimate(int fftsize, int numofFFTs, int overlap) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	size_t workSize;
	cufftSafeCall(hipfftEstimateMany( rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch,&workSize));

	cout << "cufft plan aprox buffer: " << workSize<< " bytes\n";
	return workSize;
}

//STATISTICS FUNCTIONS

void maxCompute(int numofIncoherentSums, Npp32f *deviceDataIncoherentSum, int fftsize, Npp32f *deviceArrayMaxs,
	 int *deviceArrayPos, Npp8u * pDeviceBuffer, int samplesAvoidMax) {

	for (int i = 0; i < numofIncoherentSums; i++) {

		nppsMaxIndx_32f(&deviceDataIncoherentSum[i*fftsize+(samplesAvoidMax)], fftsize+ samplesAvoidMax, &deviceArrayMaxs[i], &deviceArrayPos[i], pDeviceBuffer);
	}
}

void stdCompute(int numofIncoherentSums, Npp32f *dataIncoherentSum, int fftsize,
	Npp32f *deviceArraystd, int *arrayPos, Npp8u * pStdDeviceBuffer, int peakRange,int stdLength, Npp32f *devicearrayMean) {

	int leftPeakIndex, rightPeakIndex;
	//stdLength = (fftsize / 2) - ((peakRange) / 2)-1;
	for (int i = 0; i < numofIncoherentSums; i++) {
		
		leftPeakIndex = arrayPos[i] - peakRange/2;
		rightPeakIndex = arrayPos[i] + peakRange/2;
		
		if (rightPeakIndex >= fftsize) {//case 2
			rightPeakIndex = rightPeakIndex % fftsize;
			//stdLength = leftPeakIndex - rightPeakIndex;
			nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize+ rightPeakIndex], stdLength,&devicearrayMean[i],&deviceArraystd[i], pStdDeviceBuffer);
		}
		else if (leftPeakIndex < 0) {//case 3
			leftPeakIndex = fftsize + leftPeakIndex;
			//stdLength = leftPeakIndex-rightPeakIndex ;
			nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &devicearrayMean[i], &deviceArraystd[i], pStdDeviceBuffer);
		}
		else {//case 1
			if (arrayPos[i] < fftsize / 2) {
				//stdLength = fftsize- rightPeakIndex;
				nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &devicearrayMean[i], &deviceArraystd[i], pStdDeviceBuffer);
			}
			else {
				//stdLength = leftPeakIndex;
				nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize], stdLength, &devicearrayMean[i], &deviceArraystd[i], pStdDeviceBuffer);
			}			
		}		
	}
}

