#include "IOFunc.cuh"
//READ FUNCTIONS

void readdata(int length, int offsetFromBeg, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	//float num1,num2;

	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg * sizeof(hipfftComplex));

		myfile.read((char*)data, length * sizeof(hipfftComplex));
		/*int k = 0;
		while (k < length)
		{

			myfile.read((char*)&num1, sizeof(num1));
			myfile.read((char*)&num2, sizeof(num2));
			data[k].x = num1;
			data[k].y =  num2;
			k++;
		}*/
		myfile.close();
	}
	else cout << "Unable to open file of floats for reading " << name << "\n";
}

void readRealData(int length, int offsetFromBeg, int bytesToRead, char *data, string name)
{
	if (length > bytesToRead) {
		cout << "Error: iput length bigger than bytesToRead\n";
		exit(0);
	}

	ifstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg * sizeof(char));
		myfile.read(data, length);

		myfile.close();
		if (length < bytesToRead) {
			cout << "Warning: length smaller than bytesToRead, " << bytesToRead - length << " Bytes filled with 0 \n Last/s incoherents will be incomplete \n";
			memset(&data[length], 0, bytesToRead - length);
		}
	}
	else cout << "Unable to open file of Real Data for reading " << name << "\n";
}

void readdataInt(int length, int offsetFromBeg, __int16 *data, string name)
{
	
	ifstream myfile;
	myfile.open(name, ios::binary);
	//float num1,num2;

	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg * sizeof(__int16));

		myfile.read((char*)data, 2* length * sizeof(__int16));
		
		myfile.close();
	}
	else cout << "Unable to open file of floats for reading " << name << "\n";

}

//WRITE FUNCTIONS

void writeIncoh(int N, hipComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N / 2; ii++)
		{
			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}
		myfile.close();
	}

	else cout << "Unable to open file of incoh for writting " << name << "\n";
}

void writeMaxs(int N, Npp32f *dataMaxValue, int *dataMaxPos, Npp32f *hostarrayStd, Npp32f *hostarrayMean, int doppler, string name, int iteration, int ddmRes,
	int ddmQuant, int origIncohNum) {
	float freq, aux;
	ofstream myfile;
	myfile.open(name, ios::binary);//
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			aux = float(dataMaxPos[ii]);
			freq = doppler - (ddmRes * (ddmQuant / 2)) + ((ii / (origIncohNum))*(ddmRes));
			myfile.write((char*)&aux, sizeof(float));
			myfile.write((char*)&dataMaxValue[ii], sizeof(float));
			myfile.write((char*)&hostarrayMean[ii], sizeof(float));
			myfile.write((char*)&hostarrayStd[ii], sizeof(float));
			myfile.write((char*)&freq, sizeof(float));
			//myfile <<float( dataMaxPos[ii])<< " "<<dataMaxValue[ii] << " " << hostarrayMean[ii] << " " << hostarrayStd[ii] << " " << freq << " ";
		}
		myfile.close();
	}

	else cout << "Unable to open file of Maxs " << name << "\n";
}

void writedata(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.write((char*)data1, N * sizeof(hipfftComplex));
		/*for (int ii = 0; ii < N; ii++)
		{

			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}*/
		myfile.close();
	}

	else cout << "Unable to open file of data to write " << name << "\n";
}

void writetime(int N, string name, long long *readtime, long long *writetime, long long *looptime
	, long long *mask_elapsed_secs, long long *extenddop_elapsed_secs, long long *doppler_elapsed_secs,
	long long *fft_elapsed_secs, long long *mult_elapsed_secs, long long *ifft_elapsed_secs,
	long long *incho_elapsed_secs, long long *max_elapsed_secs, long long *savep_elapsed_secs, long long *std_elapsed_secs) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		myfile << "Atempt\t\tReadT.\t\tMask\t\tExtend\t\tDoppler\t\tFFT\t\tMul\t\tIFFT\t\tIncoh\t\tMax\t\tSaveP.\t\tSTD\t\tWriteT.\t\tLoopT." << "\n";
		for (int ii = 0; ii < N; ii++)
		{
			myfile << ii << "\t\t" << readtime[ii] << "\t\t" << mask_elapsed_secs[ii] << "\t\t"
				<< extenddop_elapsed_secs[ii] << "\t\t" << doppler_elapsed_secs[ii] << "\t\t"
				<< fft_elapsed_secs[ii] << "\t\t" << mult_elapsed_secs[ii] << "\t\t"
				<< ifft_elapsed_secs[ii] << "\t\t"
				<< incho_elapsed_secs[ii] << "\t\t" << max_elapsed_secs[ii] << "\t\t"
				<< savep_elapsed_secs[ii] << "\t\t" << std_elapsed_secs[ii] << "\t\t"
				<< writetime[ii] << "\t\t" << looptime[ii] << "\n";

		}
		myfile.close();
	}

	else cout << "Unable to open file of times " << name << "\n";
}
